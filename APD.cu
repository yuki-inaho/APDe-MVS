#include "hip/hip_runtime.h"
#include "APD.h"

__device__  void sort_small(float* d, const int n)
{
	int j;
	for (int i = 1; i < n; i++) {
		float tmp = d[i];
		for (j = i; j >= 1 && tmp < d[j - 1]; j--)
			d[j] = d[j - 1];
		d[j] = tmp;
	}
}

__device__  void sort_small(uchar* d, const int n)
{
    int j;
    for (int i = 1; i < n; i++) {
        float tmp = d[i];
        for (j = i; j >= 1 && tmp < d[j - 1]; j--)
            d[j] = d[j - 1];
        d[j] = tmp;
    }
}

__device__ void sort_small_weighted(short2* points, float* w, int n)
{
	int j;
	for (int i = 1; i < n; i++) {
		short2 tmp = points[i];
		float tmp_w = w[i];
		for (j = i; j >= 1 && tmp_w < w[j - 1]; j--) {
			points[j] = points[j - 1];
			w[j] = w[j - 1];
		}
		points[j] = tmp;
		w[j] = tmp_w;
	}
}

__device__ void getTopNIndex(const uchar *view_weight, int *top_n_index, int n)
{
    // copy view_weight to temp
    float temp[MAX_IMAGES];
    for (int i = 0; i < MAX_IMAGES; ++i) {
        temp[i] = view_weight[i];
    }
    // sort temp and get top n index
    for (int i = 0; i < n; ++i) {
        int max_index = 0;
        for (int j = 1; j < MAX_IMAGES; ++j) {
            if (temp[j] > temp[max_index]) {
                max_index = j;
            }
        }
        top_n_index[i] = max_index;
        temp[max_index] = 0;
    }
}

__device__ int FindMinCostIndex(const float* costs, const int n)
{
	float min_cost = costs[0];
	int min_cost_idx = 0;
	for (int idx = 1; idx < n; ++idx) {
		if (costs[idx] <= min_cost) {
			min_cost = costs[idx];
			min_cost_idx = idx;
		}
	}
	return min_cost_idx;
}

__device__  void setBit(unsigned int* input, const unsigned int n)
{
	(*input) |= (unsigned int)(1 << n);
}

__device__  void unSetBit(unsigned int* input, const unsigned int n)
{
	(*input) &= (unsigned int)(0xFFFFFFFE << n);
}

__device__  int isSet(unsigned int input, const unsigned int n)
{
	return (input >> n) & 1;
}

__device__ void Mat33DotVec3(const float mat[9], const float4 vec, float4* result)
{
	result->x = mat[0] * vec.x + mat[1] * vec.y + mat[2] * vec.z;
	result->y = mat[3] * vec.x + mat[4] * vec.y + mat[5] * vec.z;
	result->z = mat[6] * vec.x + mat[7] * vec.y + mat[8] * vec.z;
}

__device__ float Vec3DotVec3(const float4 vec1, const float4 vec2)
{
	return vec1.x * vec2.x + vec1.y * vec2.y + vec1.z * vec2.z;
}

__device__ float Vec3DotVec3(const float3 vec1, const float3 vec2)
{
	return vec1.x * vec2.x + vec1.y * vec2.y + vec1.z * vec2.z;
}

__device__ float3 Vec3CrossVec3(const float3 vec1, const float3 vec2)
{
	float3 cross_vec;
	cross_vec.x = vec1.y * vec2.z - vec2.y * vec1.z;
	cross_vec.y = -(vec1.x * vec2.z - vec2.x * vec1.z);
	cross_vec.z = vec1.x * vec2.y - vec2.x * vec1.y;
	return cross_vec;
}

__device__ float Vec2DotVec2(float2 a, float2 b) {
	return a.x * b.x + a.y * b.y;
}

__device__ float Vec2CrossVec2(float2 a, float2 b) {
	return a.x * b.y - a.y * b.x;
}

__device__ bool PointinTriangle(short2 A, short2 B, short2 C, int2 P)
{
	float2 AB = make_float2(B.x - A.x, B.y - A.y);
	float2 BC = make_float2(C.x - B.x, C.y - B.y);
	float2 CA = make_float2(A.x - C.x, A.y - C.y);
	float AB_ = sqrt(AB.x * AB.x + AB.y * AB.y);
	float BC_ = sqrt(BC.x * BC.x + BC.y * BC.y);
	float CA_ = sqrt(CA.x * CA.x + CA.y * CA.y);
	if (AB_ <= 2 || BC_ <= 2 || CA_ <= 2) {
		return false;
	}
	if (!(AB_ + BC_ > CA_ && BC_ + CA_ > AB_ && AB_ + CA_ > BC_)) {
		return false;
	}
	float2 PA = make_float2(A.x - P.x, A.y - P.y);
	float2 PB = make_float2(B.x - P.x, B.y - P.y);
	float2 PC = make_float2(C.x - P.x, C.y - P.y);
	float t1 = Vec2CrossVec2(PA, PB);
	float t2 = Vec2CrossVec2(PB, PC);
	float t3 = Vec2CrossVec2(PC, PA);
	return t1 * t2 >= 0 && t1 * t3 >= 0;
}

__device__ float TriangleArea(float3 A, float3 B, float3 C)
{
	float3 AB = make_float3(B.x - A.x, B.y - A.y, B.z - A.z);
	float3 BC = make_float3(C.x - B.x, C.y - B.y, C.z - B.z);
	float3 CA = make_float3(A.x - C.x, A.y - C.y, A.z - C.z);
	float AB_ = sqrt(AB.x * AB.x + AB.y * AB.y + AB.z * AB.z);
	float BC_ = sqrt(BC.x * BC.x + BC.y * BC.y + BC.z * BC.z);
	float CA_ = sqrt(CA.x * CA.x + CA.y * CA.y + CA.z * CA.z);
	float P = (AB_ + BC_ + CA_) / 2.0f;
	return sqrt(P * (P - AB_) * (P - BC_) * (P - CA_));
}

__device__ void NormalizeVec3(float4* vec)
{
	const float normSquared = vec->x * vec->x + vec->y * vec->y + vec->z * vec->z;
	const float inverse_sqrt = rsqrtf(normSquared);
	vec->x *= inverse_sqrt;
	vec->y *= inverse_sqrt;
	vec->z *= inverse_sqrt;
}

__device__ void NormalizeVec2(float2* vec)
{
	const float normSquared = vec->x * vec->x + vec->y * vec->y;
	const float inverse_sqrt = rsqrtf(normSquared);
	vec->x *= inverse_sqrt;
	vec->y *= inverse_sqrt;
}

__device__ void TransformPDFToCDF(float* probs, const int num_probs)
{
	float prob_sum = 0.0f;
	for (int i = 0; i < num_probs; ++i) {
		prob_sum += probs[i];
	}
	const float inv_prob_sum = 1.0f / prob_sum;

	float cum_prob = 0.0f;
	for (int i = 0; i < num_probs; ++i) {
		const float prob = probs[i] * inv_prob_sum;
		cum_prob += prob;
		probs[i] = cum_prob;
	}
}

__device__ void Get3DPoint(const Camera& camera, const int2 p, const float depth, float* X)
{
	X[0] = depth * (p.x - camera.K[2]) / camera.K[0];
	X[1] = depth * (p.y - camera.K[5]) / camera.K[4];
	X[2] = depth;
}

__device__ void Get3DPoint(const Camera& camera, const short2 p, const float depth, float* X)
{
	X[0] = depth * (p.x - camera.K[2]) / camera.K[0];
	X[1] = depth * (p.y - camera.K[5]) / camera.K[4];
	X[2] = depth;
}

__device__ float4 GetViewDirection(const Camera& camera, const int2 p, const float depth)
{
	float X[3];
	Get3DPoint(camera, p, depth, X);
	float norm = sqrt(X[0] * X[0] + X[1] * X[1] + X[2] * X[2]);

	float4 view_direction;
	view_direction.x = X[0] / norm;
	view_direction.y = X[1] / norm;
	view_direction.z = X[2] / norm;
	view_direction.w = 0;
	return view_direction;
}

__device__ float GetDistance2Origin(const Camera& camera, const int2 p, const float depth, const float4 normal)
{
	float X[3];
	Get3DPoint(camera, p, depth, X);
	return -(normal.x * X[0] + normal.y * X[1] + normal.z * X[2]);
}

__device__   float SpatialGauss(float x1, float y1, float x2, float y2, float sigma, float mu = 0.0)
{
	float dis = pow(x1 - x2, 2) + pow(y1 - y2, 2) - mu;
	return exp(-1.0 * dis / (2 * sigma * sigma));
}

__device__  float RangeGauss(float x, float sigma, float mu = 0.0)
{
	float x_p = x - mu;
	return exp(-1.0 * (x_p * x_p) / (2 * sigma * sigma));
}

__device__ float ComputeDepthfromPlaneHypothesis(const Camera& camera, const float4 plane_hypothesis, const int2 p)
{
	return -plane_hypothesis.w * camera.K[0] / ((p.x - camera.K[2]) * plane_hypothesis.x + (camera.K[0] / camera.K[4]) * (p.y - camera.K[5]) * plane_hypothesis.y + camera.K[0] * plane_hypothesis.z);
}

__device__ float4 GenerateRandomNormal(const Camera& camera, const int2 p, hiprandState* rand_state, const float depth)
{
	float4 normal;
	float q1 = 1.0f;
	float q2 = 1.0f;
	float s = 2.0f;
	while (s >= 1.0f) {
		q1 = 2.0f * hiprand_uniform(rand_state) - 1.0f;
		q2 = 2.0f * hiprand_uniform(rand_state) - 1.0f;
		s = q1 * q1 + q2 * q2;
	}
	const float sq = sqrt(1.0f - s);
	normal.x = 2.0f * q1 * sq;
	normal.y = 2.0f * q2 * sq;
	normal.z = 1.0f - 2.0f * s;
	normal.w = 0;

	float4 view_direction = GetViewDirection(camera, p, depth);
	float dot_product = normal.x * view_direction.x + normal.y * view_direction.y + normal.z * view_direction.z;
	if (dot_product > 0.0f) {
		normal.x = -normal.x;
		normal.y = -normal.y;
		normal.z = -normal.z;
	}
	NormalizeVec3(&normal);
	return normal;
}

__device__ float4 GeneratePerturbedNormal(const Camera& camera, const int2 p, const float4 normal, hiprandState* rand_state, const float perturbation)
{
	float4 view_direction = GetViewDirection(camera, p, 1.0f);

	const float a1 = (hiprand_uniform(rand_state) - 0.5f) * perturbation;
	const float a2 = (hiprand_uniform(rand_state) - 0.5f) * perturbation;
	const float a3 = (hiprand_uniform(rand_state) - 0.5f) * perturbation;

	const float sin_a1 = sin(a1);
	const float sin_a2 = sin(a2);
	const float sin_a3 = sin(a3);
	const float cos_a1 = cos(a1);
	const float cos_a2 = cos(a2);
	const float cos_a3 = cos(a3);

	float R[9];
	R[0] = cos_a2 * cos_a3;
	R[1] = cos_a3 * sin_a1 * sin_a2 - cos_a1 * sin_a3;
	R[2] = sin_a1 * sin_a3 + cos_a1 * cos_a3 * sin_a2;
	R[3] = cos_a2 * sin_a3;
	R[4] = cos_a1 * cos_a3 + sin_a1 * sin_a2 * sin_a3;
	R[5] = cos_a1 * sin_a2 * sin_a3 - cos_a3 * sin_a1;
	R[6] = -sin_a2;
	R[7] = cos_a2 * sin_a1;
	R[8] = cos_a1 * cos_a2;

	float4 normal_perturbed;
	Mat33DotVec3(R, normal, &normal_perturbed);

	if (Vec3DotVec3(normal_perturbed, view_direction) >= 0.0f) {
		normal_perturbed = normal;
	}

	NormalizeVec3(&normal_perturbed);
	return normal_perturbed;
}

__device__ float4 GenerateRandomPlaneHypothesis(const Camera& camera, const int2 p, hiprandState* rand_state, const float depth_min, const float depth_max)
{
	float depth = hiprand_uniform(rand_state) * (depth_max - depth_min) + depth_min;
	float4 plane_hypothesis = GenerateRandomNormal(camera, p, rand_state, depth);
	plane_hypothesis.w = GetDistance2Origin(camera, p, depth, plane_hypothesis);
	return plane_hypothesis;
}

__device__ float4 GeneratePertubedPlaneHypothesis(const Camera& camera, const int2 p, hiprandState* rand_state, const float perturbation, const float4 plane_hypothesis_now, const float depth_now, const float depth_min, const float depth_max)
{
	float depth_perturbed = depth_now;

	float dist_perturbed = plane_hypothesis_now.w;
	const float dist_min_perturbed = (1 - perturbation) * dist_perturbed;
	const float dist_max_perturbed = (1 + perturbation) * dist_perturbed;
	float4 plane_hypothesis_temp = plane_hypothesis_now;
	do {
		dist_perturbed = hiprand_uniform(rand_state) * (dist_max_perturbed - dist_min_perturbed) + dist_min_perturbed;
		plane_hypothesis_temp.w = dist_perturbed;
		depth_perturbed = ComputeDepthfromPlaneHypothesis(camera, plane_hypothesis_temp, p);
	} while (depth_perturbed < depth_min && depth_perturbed > depth_max);

	float4 plane_hypothesis = GeneratePerturbedNormal(camera, p, plane_hypothesis_now, rand_state, perturbation * M_PI);
	plane_hypothesis.w = dist_perturbed;
	return plane_hypothesis;
}

__device__ void ComputeHomography(const Camera& ref_camera, const Camera& src_camera, const float4 plane_hypothesis, float* H)
{
	float ref_C[3];
	float src_C[3];
	ref_C[0] = -(ref_camera.R[0] * ref_camera.t[0] + ref_camera.R[3] * ref_camera.t[1] + ref_camera.R[6] * ref_camera.t[2]);
	ref_C[1] = -(ref_camera.R[1] * ref_camera.t[0] + ref_camera.R[4] * ref_camera.t[1] + ref_camera.R[7] * ref_camera.t[2]);
	ref_C[2] = -(ref_camera.R[2] * ref_camera.t[0] + ref_camera.R[5] * ref_camera.t[1] + ref_camera.R[8] * ref_camera.t[2]);
	src_C[0] = -(src_camera.R[0] * src_camera.t[0] + src_camera.R[3] * src_camera.t[1] + src_camera.R[6] * src_camera.t[2]);
	src_C[1] = -(src_camera.R[1] * src_camera.t[0] + src_camera.R[4] * src_camera.t[1] + src_camera.R[7] * src_camera.t[2]);
	src_C[2] = -(src_camera.R[2] * src_camera.t[0] + src_camera.R[5] * src_camera.t[1] + src_camera.R[8] * src_camera.t[2]);

	float R_relative[9];
	float C_relative[3];
	float t_relative[3];
	R_relative[0] = src_camera.R[0] * ref_camera.R[0] + src_camera.R[1] * ref_camera.R[1] + src_camera.R[2] * ref_camera.R[2];
	R_relative[1] = src_camera.R[0] * ref_camera.R[3] + src_camera.R[1] * ref_camera.R[4] + src_camera.R[2] * ref_camera.R[5];
	R_relative[2] = src_camera.R[0] * ref_camera.R[6] + src_camera.R[1] * ref_camera.R[7] + src_camera.R[2] * ref_camera.R[8];
	R_relative[3] = src_camera.R[3] * ref_camera.R[0] + src_camera.R[4] * ref_camera.R[1] + src_camera.R[5] * ref_camera.R[2];
	R_relative[4] = src_camera.R[3] * ref_camera.R[3] + src_camera.R[4] * ref_camera.R[4] + src_camera.R[5] * ref_camera.R[5];
	R_relative[5] = src_camera.R[3] * ref_camera.R[6] + src_camera.R[4] * ref_camera.R[7] + src_camera.R[5] * ref_camera.R[8];
	R_relative[6] = src_camera.R[6] * ref_camera.R[0] + src_camera.R[7] * ref_camera.R[1] + src_camera.R[8] * ref_camera.R[2];
	R_relative[7] = src_camera.R[6] * ref_camera.R[3] + src_camera.R[7] * ref_camera.R[4] + src_camera.R[8] * ref_camera.R[5];
	R_relative[8] = src_camera.R[6] * ref_camera.R[6] + src_camera.R[7] * ref_camera.R[7] + src_camera.R[8] * ref_camera.R[8];
	C_relative[0] = (ref_C[0] - src_C[0]);
	C_relative[1] = (ref_C[1] - src_C[1]);
	C_relative[2] = (ref_C[2] - src_C[2]);
	t_relative[0] = src_camera.R[0] * C_relative[0] + src_camera.R[1] * C_relative[1] + src_camera.R[2] * C_relative[2];
	t_relative[1] = src_camera.R[3] * C_relative[0] + src_camera.R[4] * C_relative[1] + src_camera.R[5] * C_relative[2];
	t_relative[2] = src_camera.R[6] * C_relative[0] + src_camera.R[7] * C_relative[1] + src_camera.R[8] * C_relative[2];

	H[0] = R_relative[0] - t_relative[0] * plane_hypothesis.x / plane_hypothesis.w;
	H[1] = R_relative[1] - t_relative[0] * plane_hypothesis.y / plane_hypothesis.w;
	H[2] = R_relative[2] - t_relative[0] * plane_hypothesis.z / plane_hypothesis.w;
	H[3] = R_relative[3] - t_relative[1] * plane_hypothesis.x / plane_hypothesis.w;
	H[4] = R_relative[4] - t_relative[1] * plane_hypothesis.y / plane_hypothesis.w;
	H[5] = R_relative[5] - t_relative[1] * plane_hypothesis.z / plane_hypothesis.w;
	H[6] = R_relative[6] - t_relative[2] * plane_hypothesis.x / plane_hypothesis.w;
	H[7] = R_relative[7] - t_relative[2] * plane_hypothesis.y / plane_hypothesis.w;
	H[8] = R_relative[8] - t_relative[2] * plane_hypothesis.z / plane_hypothesis.w;

	float tmp[9];
	tmp[0] = H[0] / ref_camera.K[0];
	tmp[1] = H[1] / ref_camera.K[4];
	tmp[2] = -H[0] * ref_camera.K[2] / ref_camera.K[0] - H[1] * ref_camera.K[5] / ref_camera.K[4] + H[2];
	tmp[3] = H[3] / ref_camera.K[0];
	tmp[4] = H[4] / ref_camera.K[4];
	tmp[5] = -H[3] * ref_camera.K[2] / ref_camera.K[0] - H[4] * ref_camera.K[5] / ref_camera.K[4] + H[5];
	tmp[6] = H[6] / ref_camera.K[0];
	tmp[7] = H[7] / ref_camera.K[4];
	tmp[8] = -H[6] * ref_camera.K[2] / ref_camera.K[0] - H[7] * ref_camera.K[5] / ref_camera.K[4] + H[8];

	H[0] = src_camera.K[0] * tmp[0] + src_camera.K[2] * tmp[6];
	H[1] = src_camera.K[0] * tmp[1] + src_camera.K[2] * tmp[7];
	H[2] = src_camera.K[0] * tmp[2] + src_camera.K[2] * tmp[8];
	H[3] = src_camera.K[4] * tmp[3] + src_camera.K[5] * tmp[6];
	H[4] = src_camera.K[4] * tmp[4] + src_camera.K[5] * tmp[7];
	H[5] = src_camera.K[4] * tmp[5] + src_camera.K[5] * tmp[8];
	H[6] = src_camera.K[8] * tmp[6];
	H[7] = src_camera.K[8] * tmp[7];
	H[8] = src_camera.K[8] * tmp[8];
}

__device__ float2 ComputeCorrespondingPoint(const float* H, const int2 p)
{
	float3 pt;
	pt.x = H[0] * p.x + H[1] * p.y + H[2];
	pt.y = H[3] * p.x + H[4] * p.y + H[5];
	pt.z = H[6] * p.x + H[7] * p.y + H[8];
	return make_float2(pt.x / pt.z, pt.y / pt.z);
}

__device__ float4 TransformNormal(const Camera& camera, float4 plane_hypothesis)
{
	float4 transformed_normal;
	transformed_normal.x = camera.R[0] * plane_hypothesis.x + camera.R[3] * plane_hypothesis.y + camera.R[6] * plane_hypothesis.z;
	transformed_normal.y = camera.R[1] * plane_hypothesis.x + camera.R[4] * plane_hypothesis.y + camera.R[7] * plane_hypothesis.z;
	transformed_normal.z = camera.R[2] * plane_hypothesis.x + camera.R[5] * plane_hypothesis.y + camera.R[8] * plane_hypothesis.z;
	transformed_normal.w = plane_hypothesis.w;
	return transformed_normal;
}

__device__ float4 TransformNormal2RefCam(const Camera& camera, float4 plane_hypothesis)
{
	float4 transformed_normal;
	transformed_normal.x = camera.R[0] * plane_hypothesis.x + camera.R[1] * plane_hypothesis.y + camera.R[2] * plane_hypothesis.z;
	transformed_normal.y = camera.R[3] * plane_hypothesis.x + camera.R[4] * plane_hypothesis.y + camera.R[5] * plane_hypothesis.z;
	transformed_normal.z = camera.R[6] * plane_hypothesis.x + camera.R[7] * plane_hypothesis.y + camera.R[8] * plane_hypothesis.z;
	transformed_normal.w = plane_hypothesis.w;
	return transformed_normal;
}

__device__ short2 GetAnchorPoint(const int2 p, const int index, const DataPassHelper* helper) {
	const unsigned offset = helper->anchors_map_cuda[p.x + p.y * helper->width] * ANCHOR_NUM;
	short2 anchor_pt = helper->anchors_cuda[offset + index];
	return anchor_pt;
}

__device__ void Softmax(float *costs, int n) {
    float max_cost = -1e10;
    for (int i = 0; i < n; i++) {
        if (costs[i] > max_cost) {
            max_cost = costs[i];
        }
    }
    float sum = 0;
    for (int i = 0; i < n; i++) {
        costs[i] = exp(costs[i] - max_cost);
        sum += costs[i];
    }
    for (int i = 0; i < n; i++) {
        costs[i] /= sum;
    }
}

__device__ float ComputeBilateralNCCNew(
        const int2 p,
        const int src_idx,
        const float4 plane_hypothesis,
        const DataPassHelper* helper
) {
    const hipTextureObject_t ref_image = helper->texture_objects_cuda[0].images[0];
    const Camera& ref_camera = helper->cameras_cuda[0];
    const hipTextureObject_t src_image = helper->texture_objects_cuda[0].images[src_idx];
    const Camera& src_camera = helper->cameras_cuda[src_idx];
    const PatchMatchParams* params = helper->params;
    const uchar* weak_info = helper->weak_info_cuda;
    const int width = helper->width;
    const int height = helper->height;
    const int center = p.x + p.y * width;
    const uchar *sa_mask = helper->sa_mask_cuda;
    const uchar center_sa_id = sa_mask[center];
    bool use_sa_mask = (sa_mask[center] != 0);


    const float cost_max = 2.0f;

    float H[9];
    ComputeHomography(ref_camera, src_camera, plane_hypothesis, H);
    float2 pt = ComputeCorrespondingPoint(H, p);
    if (pt.x >= src_camera.width || pt.x < 0.0f || pt.y >= src_camera.height || pt.y < 0.0f) {
        return cost_max;
    }

    float cost = 0.0f;
    float strong_costs[9];
    int strong_costs_num = 0;

    if (weak_info[center] == WEAK) {
        // for weak texture area use deformable ncc
        const float ref_center_pix = tex2D<float>(ref_image, p.x + 0.5f, p.y + 0.5f);
        // the strong points
        float center_cost = 0.0f;
        float strong_cost = 0.0f;
        float strong_weight = 0;
        for (int k = 0; k < ANCHOR_NUM; ++k) {
            const short2 anchor_pt = GetAnchorPoint(p, k, helper);
            if (anchor_pt.x == -1 || anchor_pt.y == -1) {
                continue;
            }
            if (use_sa_mask) {
                if (sa_mask[anchor_pt.x + anchor_pt.y * width] != center_sa_id) {
                    continue;
                }
            }

            float2 anchor_src_pt = ComputeCorrespondingPoint(H, make_int2(anchor_pt.x, anchor_pt.y));
            if (anchor_src_pt.x < 0 || anchor_src_pt.y < 0 || anchor_src_pt.x >= width || anchor_src_pt.y >= height) {
                if (k != 0) {
                    unsigned int view_info = helper->selected_views_cuda[anchor_pt.x + anchor_pt.y * width];
                    if (isSet(view_info, src_idx - 1)) {
                        strong_costs[strong_costs_num++] = cost_max;
                        strong_weight += 1;
                    }
                    continue;
                }
                else {
                    return cost_max;
                }
            }
            // compute ncc for this point
            float sum_ref = 0.0f;
            float sum_ref_ref = 0.0f;
            float sum_src = 0.0f;
            float sum_src_src = 0.0f;
            float sum_ref_src = 0.0f;
            float bilateral_weight_sum = 0.0f;
            const float ref_center_pix = tex2D<float>(ref_image, p.x + 0.5f, p.y + 0.5f);
            int radius = (k == 0 ? params->strong_radius : params->weak_radius);
            int increment = (k == 0 ? params->strong_increment : params->weak_increment);
            for (int i = -radius; i <= radius; i += increment) {
                for (int j = -radius; j <= radius; j += increment) {
                    const int2 ref_pt = make_int2(anchor_pt.x + i, anchor_pt.y + j);
                    if (use_sa_mask) {
                        if (sa_mask[ref_pt.x + ref_pt.y * width] != center_sa_id) {
                            continue;
                        }
                    }
                    const float ref_pix = tex2D<float>(ref_image, ref_pt.x + 0.5f, ref_pt.y + 0.5f);
                    float2 src_pt = ComputeCorrespondingPoint(H, ref_pt);
                    const float src_pix = tex2D<float>(src_image, src_pt.x + 0.5f, src_pt.y + 0.5f);
                    float weight = 1.0f;
                    sum_ref += weight * ref_pix;
                    sum_ref_ref += weight * ref_pix * ref_pix;
                    sum_src += weight * src_pix;
                    sum_src_src += weight * src_pix * src_pix;
                    sum_ref_src += weight * ref_pix * src_pix;
                    bilateral_weight_sum += weight;
                }
            }
            if (bilateral_weight_sum == 0.0f) {
                continue;
            }
            const float inv_bilateral_weight_sum = 1.0f / bilateral_weight_sum;
            sum_ref *= inv_bilateral_weight_sum;
            sum_ref_ref *= inv_bilateral_weight_sum;
            sum_src *= inv_bilateral_weight_sum;
            sum_src_src *= inv_bilateral_weight_sum;
            sum_ref_src *= inv_bilateral_weight_sum;
            const float var_ref = sum_ref_ref - sum_ref * sum_ref;
            const float var_src = sum_src_src - sum_src * sum_src;
            const float kMinVar = 1e-5f;
            float temp_cost = 0.0f;
            if (var_ref < kMinVar || var_src < kMinVar) {
                temp_cost = cost_max;
            }
            else {
                const float covar_src_ref = sum_ref_src - sum_ref * sum_src;
                const float var_ref_src = sqrt(var_ref * var_src);
                temp_cost = max(0.0f, min(cost_max, 1.0f - covar_src_ref / var_ref_src));
            }
            if (k == 0) {
                center_cost = temp_cost;
            }
            else {
                strong_costs[strong_costs_num++] = temp_cost;
                strong_weight += 1;
            }
        }
        if (strong_weight <= 1e-6f) {
            cost = center_cost;
        }
        else {
            float strong_costs_weight[9];
            for (int i = 0; i < strong_costs_num; ++i) {
                strong_costs_weight[i] = strong_costs[i];
            }
            Softmax(strong_costs_weight, strong_costs_num);
            strong_cost = 0.0f;
            for (int i = 0; i < strong_costs_num; ++i) {
                strong_cost += strong_costs_weight[i] * strong_costs[i];
            }
            strong_cost = MIN(strong_cost, cost_max);
            cost = 0.25 * center_cost + 0.75 * strong_cost;
        }
    }
    else {
        printf("error\n");
    }
    return cost;
}


__device__ float ComputeBilateralNCCOld(
	const int2 p,
	const int src_idx,
	const float4 plane_hypothesis,
	const DataPassHelper* helper
) {
	const hipTextureObject_t ref_image = helper->texture_objects_cuda[0].images[0];
	const Camera& ref_camera = helper->cameras_cuda[0];
	const hipTextureObject_t src_image = helper->texture_objects_cuda[0].images[src_idx];
	const Camera& src_camera = helper->cameras_cuda[src_idx];
    const uchar *sa_mask = helper->sa_mask_cuda;
    const uchar center_sa_id = sa_mask[p.x + p.y * ref_camera.width];

	const float cost_max = 2.0f;

	float H[9];
	ComputeHomography(ref_camera, src_camera, plane_hypothesis, H);
	float2 pt = ComputeCorrespondingPoint(H, p);
	if (pt.x >= src_camera.width || pt.x < 0.0f || pt.y >= src_camera.height || pt.y < 0.0f) {
		return cost_max;
	}
    const int radius = helper->params->strong_radius;
	const int increment = helper->params->strong_increment;
    const int center = pt.y * src_camera.width + pt.x;
	float cost = 0.0f;
    if (sa_mask[center] == 0) {
		float sum_ref = 0.0f;
		float sum_ref_ref = 0.0f;
		float sum_src = 0.0f;
		float sum_src_src = 0.0f;
		float sum_ref_src = 0.0f;
		float bilateral_weight_sum = 0.0f;

		for (int i = -radius; i <= radius; i += increment) {
			for (int j = -radius; j <= radius; j += increment) {
				const int2 ref_pt = make_int2(p.x + i, p.y + j);
				const float ref_pix = tex2D<float>(ref_image, ref_pt.x + 0.5f, ref_pt.y + 0.5f);
				float2 src_pt = ComputeCorrespondingPoint(H, ref_pt);
				const float src_pix = tex2D<float>(src_image, src_pt.x + 0.5f, src_pt.y + 0.5f);
				float weight = 1.0f;
				sum_ref += weight * ref_pix;
				sum_ref_ref += weight * ref_pix * ref_pix;
				sum_src += weight * src_pix;
				sum_src_src += weight * src_pix * src_pix;
				sum_ref_src += weight * ref_pix * src_pix;
				bilateral_weight_sum += weight;
			}
        }
		const float inv_bilateral_weight_sum = 1.0f / bilateral_weight_sum;
		sum_ref *= inv_bilateral_weight_sum;
		sum_ref_ref *= inv_bilateral_weight_sum;
		sum_src *= inv_bilateral_weight_sum;
		sum_src_src *= inv_bilateral_weight_sum;
		sum_ref_src *= inv_bilateral_weight_sum;

		const float var_ref = sum_ref_ref - sum_ref * sum_ref;
		const float var_src = sum_src_src - sum_src * sum_src;

		const float kMinVar = 1e-5f;
		if (var_ref < kMinVar || var_src < kMinVar) {
			cost = cost_max;
		}
		else {
			const float covar_src_ref = sum_ref_src - sum_ref * sum_src;
			const float var_ref_src = sqrt(var_ref * var_src);
			cost = max(0.0f, min(cost_max, 1.0f - covar_src_ref / var_ref_src));
		}
	}
    else {
        const int sign[] = {1, 1, -1, -1, 1, -1, -1, 1};
        const int offset[] = {1, 1, 3, 1, 1, 3, 1, 5, 3, 3, 5, 1, 5, 3, 3, 5, 5, 5};
        float sum_ref = 0.0f;
        float sum_ref_ref = 0.0f;
        float sum_src = 0.0f;
        float sum_src_src = 0.0f;
        float sum_ref_src = 0.0f;
        float bilateral_weight_sum = 0.0f;
        for (int i = 0; i < 4; ++i) {
            const int x_sign = sign[i * 2];
            const int y_sign = sign[i * 2 + 1];
            for (int j = 0; j < 9; ++j) {
                const int x_offset = offset[j * 2] * x_sign;
                const int y_offset = offset[j * 2 + 1] * y_sign;
                const int2 ref_pt = make_int2(p.x + x_offset, p.y + y_offset);
                if (ref_pt.x < 0 || ref_pt.x >= ref_camera.width || ref_pt.y < 0 || ref_pt.y >= ref_camera.height) {
                    continue;
                }
                const int ref_idx = ref_pt.y * ref_camera.width + ref_pt.x;
                if (sa_mask[ref_idx] != center_sa_id) {
                    break;
                }
                const float ref_pix = tex2D<float>(ref_image, ref_pt.x + 0.5f, ref_pt.y + 0.5f);
                float2 src_pt = ComputeCorrespondingPoint(H, ref_pt);
                const float src_pix = tex2D<float>(src_image, src_pt.x + 0.5f, src_pt.y + 0.5f);
                float weight = 1.0f;
                sum_ref += weight * ref_pix;
                sum_ref_ref += weight * ref_pix * ref_pix;
                sum_src += weight * src_pix;
                sum_src_src += weight * src_pix * src_pix;
                sum_ref_src += weight * ref_pix * src_pix;
                bilateral_weight_sum += weight;
            }
        }
        const float inv_bilateral_weight_sum = 1.0f / bilateral_weight_sum;
        sum_ref *= inv_bilateral_weight_sum;
        sum_ref_ref *= inv_bilateral_weight_sum;
        sum_src *= inv_bilateral_weight_sum;
        sum_src_src *= inv_bilateral_weight_sum;
        sum_ref_src *= inv_bilateral_weight_sum;

        const float var_ref = sum_ref_ref - sum_ref * sum_ref;
        const float var_src = sum_src_src - sum_src * sum_src;

        const float kMinVar = 1e-5f;
        if (var_ref < kMinVar || var_src < kMinVar) {
            cost = cost_max;
        }
        else {
            const float covar_src_ref = sum_ref_src - sum_ref * sum_src;
            const float var_ref_src = sqrt(var_ref * var_src);
            cost = max(0.0f, min(cost_max, 1.0f - covar_src_ref / var_ref_src));
        }

    }
	return cost;
}

__device__ float ComputeMultiViewInitialCostandSelectedViews(
	const int2 p,
	DataPassHelper* helper
) {
	PatchMatchParams* params = helper->params;
	unsigned int* selected_views = helper->selected_views_cuda;
	int center = p.x + p.y * helper->width;
	float4 plane_hypothesis = helper->plane_hypotheses_cuda[center];
	uchar *weak_info = helper->weak_info_cuda;
	
	float cost_max = 2.0f;
	float cost_vector[32] = { 2.0f };
	float cost_vector_copy[32] = { 2.0f };
	int cost_count = 0;
	int num_valid_views = 0;

	for (int i = 1; i < params->num_images; ++i) {
		float c = 0.0f;
		if (params->use_APD && weak_info[center] == WEAK) {
			c = ComputeBilateralNCCNew(p, i, plane_hypothesis, helper);
		} else {
			c = ComputeBilateralNCCOld(p, i, plane_hypothesis, helper);
		}
		cost_vector[i - 1] = c;
		cost_vector_copy[i - 1] = c;
		cost_count++;
		if (c < cost_max) {
			num_valid_views++;
		}
	}

	sort_small(cost_vector, cost_count);
	selected_views[center] = 0;

	int top_k = min(num_valid_views, params->top_k);
	if (top_k > 0) {
		float cost = 0.0f;
		for (int i = 0; i < top_k; ++i) {
			cost += cost_vector[i];
		}
		float cost_threshold = cost_vector[top_k - 1];
		for (int i = 0; i < params->num_images - 1; ++i) {
			if (cost_vector_copy[i] <= cost_threshold) {
				setBit(&(selected_views[center]), i);
			}
		}
		return cost / top_k;
	}
	else {
		return cost_max;
	}
}

__device__ float ComputeMultiViewInitialCost(
	const int2 p,
	DataPassHelper* helper
) {
	PatchMatchParams* params = helper->params;
	unsigned int* selected_views = helper->selected_views_cuda;
	int center = p.x + p.y * helper->width;
	float4 plane_hypothesis = helper->plane_hypotheses_cuda[center];

	const float cost_max = 2.0f;
	int cost_count = 0;
	float cost = 0.0f;

	for (int i = 1; i < params->num_images; ++i) {
		if (isSet(selected_views[center], i - 1)) {
			float c = ComputeBilateralNCCOld(p, i, plane_hypothesis, helper);
			if (c < cost_max) {
				cost_count++;
				cost += c;
			}
			else {
				unSetBit(&(selected_views[center]), i - 1);
			}
		}
	}
	if (cost_count == 0) {
		return cost_max;
	}
	else {
		return cost / cost_count;
	}
}

__device__ void ComputeMultiViewCostVectorNew(
	const int2 p,
	float4 plane_hypothesis,
	float* cost_vector,
	DataPassHelper* helper
) {
	for (int i = 1; i < helper->params->num_images; ++i) {
		cost_vector[i - 1] = ComputeBilateralNCCNew(p, i, plane_hypothesis, helper);
	}
}

__device__ void ComputeMultiViewCostVectorOld(
	const int2 p,
	float4 plane_hypothesis,
	float* cost_vector,
	DataPassHelper* helper
) {
	for (int i = 1; i < helper->params->num_images; ++i) {
		cost_vector[i - 1] = ComputeBilateralNCCOld(p, i, plane_hypothesis, helper);
	}
}

__device__ float3 Get3DPointonWorld_cu(const float x, const float y, const float depth, const Camera& camera)
{
	float3 pointX;
	float3 tmpX;
	// Reprojection
	pointX.x = depth * (x - camera.K[2]) / camera.K[0];
	pointX.y = depth * (y - camera.K[5]) / camera.K[4];
	pointX.z = depth;

	// Rotation
	tmpX.x = camera.R[0] * pointX.x + camera.R[3] * pointX.y + camera.R[6] * pointX.z;
	tmpX.y = camera.R[1] * pointX.x + camera.R[4] * pointX.y + camera.R[7] * pointX.z;
	tmpX.z = camera.R[2] * pointX.x + camera.R[5] * pointX.y + camera.R[8] * pointX.z;

	// Transformation
	pointX.x = tmpX.x + camera.c[0];
	pointX.y = tmpX.y + camera.c[1];
	pointX.z = tmpX.z + camera.c[2];

	return pointX;
}

__device__ void ProjectonCamera_cu(const float3 PointX, const Camera& camera, float2& point, float& depth)
{
	float3 tmp;
	tmp.x = camera.R[0] * PointX.x + camera.R[1] * PointX.y + camera.R[2] * PointX.z + camera.t[0];
	tmp.y = camera.R[3] * PointX.x + camera.R[4] * PointX.y + camera.R[5] * PointX.z + camera.t[1];
	tmp.z = camera.R[6] * PointX.x + camera.R[7] * PointX.y + camera.R[8] * PointX.z + camera.t[2];

	depth = camera.K[6] * tmp.x + camera.K[7] * tmp.y + camera.K[8] * tmp.z;
	point.x = (camera.K[0] * tmp.x + camera.K[1] * tmp.y + camera.K[2] * tmp.z) / depth;
	point.y = (camera.K[3] * tmp.x + camera.K[4] * tmp.y + camera.K[5] * tmp.z) / depth;
}

__device__ float ComputeGeomConsistencyCost(
	const int2 p,
	const int src_idx,
	const float4 plane_hypothesis,
	DataPassHelper* helper
) {
	const Camera& ref_camera = helper->cameras_cuda[0];
	const Camera& src_camera = helper->cameras_cuda[src_idx];
	const hipTextureObject_t depth_image = helper->texture_depths_cuda[0].images[src_idx];

	const float max_cost = 3.0f;

	float center_cost = 0.0f;
	{
		float depth = ComputeDepthfromPlaneHypothesis(ref_camera, plane_hypothesis, p);
		float3 forward_point = Get3DPointonWorld_cu(p.x, p.y, depth, ref_camera);

		float2 src_pt;
		float src_d;
		ProjectonCamera_cu(forward_point, src_camera, src_pt, src_d);
		const float src_depth = tex2D<float>(depth_image, (int)src_pt.x + 0.5f, (int)src_pt.y + 0.5f);

		if (src_depth == 0.0f) {
			return max_cost;
		}

		float3 src_3D_pt = Get3DPointonWorld_cu(src_pt.x, src_pt.y, src_depth, src_camera);

		float2 backward_point;
		float ref_d;
		ProjectonCamera_cu(src_3D_pt, ref_camera, backward_point, ref_d);

		const float diff_col = p.x - backward_point.x;
		const float diff_row = p.y - backward_point.y;
		center_cost = sqrt(diff_col * diff_col + diff_row * diff_row);
	}
	return min(max_cost, center_cost);
}

__global__ void InitRandomStates(
	DataPassHelper* helper
) {
	const int width = helper->width;
	const int height = helper->height;
	hiprandState* rand_states = helper->rand_states_cuda;

	const int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
	if (p.x >= width || p.y >= height) {
		return;
	}
	const int center = p.y * width + p.x;
	hiprand_init(clock64(), p.y, p.x, &rand_states[center]);
}

__global__ void RandomInitialization(
	DataPassHelper* helper
) {
	int width = helper->width;
	int height = helper->height;
	const int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
	if (p.x >= width || p.y >= height) {
		return;
	}
	const int center = p.y * width + p.x;
	Camera* cameras = helper->cameras_cuda;
	float4* plane_hypotheses = helper->plane_hypotheses_cuda;
	float* costs = helper->costs_cuda;
	hiprandState* rand_states = helper->rand_states_cuda;
	PatchMatchParams* params = helper->params;

	if (params->state == FIRST_INIT) {
		plane_hypotheses[center] = GenerateRandomPlaneHypothesis(cameras[0], p, &rand_states[center], params->depth_min, params->depth_max);
		costs[center] = ComputeMultiViewInitialCostandSelectedViews(p, helper);
	}
	else {
		float4 plane_hypothesis;
		plane_hypothesis = plane_hypotheses[center];
		plane_hypothesis = TransformNormal2RefCam(cameras[0], plane_hypothesis);
		float depth = plane_hypothesis.w;
		plane_hypothesis.w = GetDistance2Origin(cameras[0], p, depth, plane_hypothesis);
		plane_hypotheses[center] = plane_hypothesis;
		costs[center] = ComputeMultiViewInitialCostandSelectedViews(p, helper);
	}
}

__device__ void PlaneHypothesisRefinementStrong(
	float4* plane_hypothesis,
	float* depth,
	float* cost,
	hiprandState* rand_state,
	const uchar* view_weights,
	const float weight_norm,
	const int2 p,
	DataPassHelper* helper

) {
	float depth_perturbation = 0.02f;
	float normal_perturbation = 0.02f;
	const Camera* cameras = helper->cameras_cuda;
	const PatchMatchParams* params = helper->params;
	float depth_min = params->depth_min;
	float depth_max = params->depth_max;

	float depth_rand = hiprand_uniform(rand_state) * (depth_max - depth_min) + depth_min;
	float4 plane_hypothesis_rand = GenerateRandomNormal(cameras[0], p, rand_state, *depth);
	float depth_perturbed = *depth;
	const float depth_min_perturbed = (1 - depth_perturbation) * depth_perturbed;
	const float depth_max_perturbed = (1 + depth_perturbation) * depth_perturbed;
	do {
		depth_perturbed = hiprand_uniform(rand_state) * (depth_max_perturbed - depth_min_perturbed) + depth_min_perturbed;
	} while (depth_perturbed < depth_min && depth_perturbed > depth_max);
	float4 plane_hypothesis_perturbed = GeneratePerturbedNormal(cameras[0], p, *plane_hypothesis, rand_state, normal_perturbation * M_PI);

	const int num_planes = 5;
	float depths[num_planes] = { depth_rand, *depth, depth_rand, *depth, depth_perturbed };
	float4 normals[num_planes] = { *plane_hypothesis, plane_hypothesis_rand, plane_hypothesis_rand, plane_hypothesis_perturbed, *plane_hypothesis };

	for (int i = 0; i < num_planes; ++i) {
		float cost_vector[32] = { 2.0f };
		float4 temp_plane_hypothesis = normals[i];
		temp_plane_hypothesis.w = GetDistance2Origin(cameras[0], p, depths[i], temp_plane_hypothesis);
		ComputeMultiViewCostVectorOld(p, temp_plane_hypothesis, cost_vector, helper);

		float temp_cost = 0.0f;
		for (int j = 0; j < params->num_images - 1; ++j) {
			if (params->geom_consistency) {
				temp_cost += view_weights[j] * (cost_vector[j] + params->geom_factor * ComputeGeomConsistencyCost(p, j + 1, temp_plane_hypothesis, helper));
			}
			else {
				temp_cost += view_weights[j] * cost_vector[j];
			}
		}
		temp_cost /= weight_norm;

		float depth_before = ComputeDepthfromPlaneHypothesis(cameras[0], temp_plane_hypothesis, p);
		if (depth_before >= depth_min && depth_before <= depth_max && temp_cost < *cost) {
			*depth = depth_before;
			*plane_hypothesis = temp_plane_hypothesis;
			*cost = temp_cost;
		}
	}
}

__device__ void PlaneHypothesisRefinementWeak(
	float4* plane_hypothesis,
	float* depth,
	float* cost,
	hiprandState* rand_state,
	const uchar* view_weights,
	const float weight_norm,
	const int2 p,
	DataPassHelper* helper

) {
	float depth_perturbation = 0.02f;
	float normal_perturbation = 0.02f;
	const Camera* cameras = helper->cameras_cuda;
	const PatchMatchParams* params = helper->params;
	float depth_min = params->depth_min;
	float depth_max = params->depth_max;
	const int center = p.x + p.y * helper->width;
	{   // test the fit plane
		float4 fit_plane_hypothesis = helper->fit_plane_hypotheses_cuda[center];
		if (fit_plane_hypothesis.x == 0 && fit_plane_hypothesis.y == 0 && fit_plane_hypothesis.z == 0) {
			return;
		}
		float cost_vector[32] = { 2.0f };
		ComputeMultiViewCostVectorNew(p, fit_plane_hypothesis, cost_vector, helper);
		float temp_cost = 0.0f;
		for (int j = 0; j < params->num_images - 1; ++j) {
			if (view_weights[j] > 0) {
				if (params->geom_consistency) {
					temp_cost += view_weights[j] * (cost_vector[j] + params->geom_factor * ComputeGeomConsistencyCost(p, j + 1, fit_plane_hypothesis, helper));
				}
				else {
					temp_cost += view_weights[j] * cost_vector[j];
				}
			}
		}
		temp_cost /= weight_norm;

		float depth_before = ComputeDepthfromPlaneHypothesis(cameras[0], fit_plane_hypothesis, p);
		if (depth_before >= depth_min && depth_before <= depth_max && temp_cost < *cost) {
			*depth = depth_before;
			*plane_hypothesis = fit_plane_hypothesis;
			*cost = temp_cost;
		}
	}
	// random refine
	{
		float depth_rand = hiprand_uniform(rand_state) * (depth_max - depth_min) + depth_min;
		float4 plane_hypothesis_rand = GenerateRandomNormal(cameras[0], p, rand_state, *depth);
		float depth_perturbed = *depth;
		const float depth_min_perturbed = (1 - depth_perturbation) * depth_perturbed;
		const float depth_max_perturbed = (1 + depth_perturbation) * depth_perturbed;
		do {
			depth_perturbed = hiprand_uniform(rand_state) * (depth_max_perturbed - depth_min_perturbed) + depth_min_perturbed;
		} while (depth_perturbed < depth_min && depth_perturbed > depth_max);
		float4 plane_hypothesis_perturbed = GeneratePerturbedNormal(cameras[0], p, *plane_hypothesis, rand_state, normal_perturbation * M_PI);

		const int num_planes = 5;
		float depths[num_planes] = { depth_rand, *depth, depth_rand, *depth, depth_perturbed };
		float4 normals[num_planes] = { *plane_hypothesis, plane_hypothesis_rand, plane_hypothesis_rand, plane_hypothesis_perturbed, *plane_hypothesis };

		for (int i = 0; i < num_planes; ++i) {
			float cost_vector[32] = { 2.0f };
			float4 temp_plane_hypothesis = normals[i];
			temp_plane_hypothesis.w = GetDistance2Origin(cameras[0], p, depths[i], temp_plane_hypothesis);
			ComputeMultiViewCostVectorNew(p, temp_plane_hypothesis, cost_vector, helper);

			float temp_cost = 0.0f;
			for (int j = 0; j < params->num_images - 1; ++j) {
				if (view_weights[j] > 0) {
					if (params->geom_consistency) {
						temp_cost += view_weights[j] * (cost_vector[j] + params->geom_factor * ComputeGeomConsistencyCost(p, j + 1, temp_plane_hypothesis, helper));
					}
					else {
						temp_cost += view_weights[j] * cost_vector[j];
					}
				}
			}
			temp_cost /= weight_norm;

			float depth_before = ComputeDepthfromPlaneHypothesis(cameras[0], temp_plane_hypothesis, p);
			if (depth_before >= depth_min && depth_before <= depth_max && temp_cost < *cost) {
				*depth = depth_before;
				*plane_hypothesis = temp_plane_hypothesis;
				*cost = temp_cost;
			}
		}
	}
}

__device__ void CheckerboardPropagationStrong(
	const int2 p,
	const int iter,
	DataPassHelper* helper
) {
	const int width = helper->width;
	const int height = helper->height;
	float4* plane_hypotheses = helper->plane_hypotheses_cuda;
	float* costs = helper->costs_cuda;
	hiprandState* rand_states = helper->rand_states_cuda;
	unsigned int* selected_views = helper->selected_views_cuda;
	PatchMatchParams* params = helper->params;
	const Camera* cameras = helper->cameras_cuda;
	int num_images = params->num_images;

	if (p.x >= width || p.y >= height) {
		return;
	}

	const int center = p.y * width + p.x;

	// Adaptive Checkerboard Sampling
	float cost_array[8][32] = { 2.0f };
	bool flag[8] = { false };
	int num_valid_pixels = 0;

	float costMin;
	int costMinPoint;

	int left_near = center - 1;
	int left_far = center - 3;
	int right_near = center + 1;
	int right_far = center + 3;
	int up_near = center - width;
	int up_far = center - 3 * width;
	int down_near = center + width;
	int down_far = center + 3 * width;
	// 0 -- up_near, 1 -- up_far, 2 -- down_near, 3 -- down_far, 4 -- left_near, 5 -- left_far, 6 -- right_near, 7 -- right_far
	// up_far
	if (p.y > 2) {
		flag[1] = true;
		num_valid_pixels++;
		costMin = costs[up_far];
		costMinPoint = up_far;
		for (int i = 1; i < 11; ++i) {
			if (p.y > 2 + 2 * i) {
				int pointTemp = up_far - 2 * i * width;
				if (costs[pointTemp] < costMin) {
					costMin = costs[pointTemp];
					costMinPoint = pointTemp;
				}
			}
		}
		up_far = costMinPoint;
		ComputeMultiViewCostVectorOld(p, plane_hypotheses[up_far], cost_array[1], helper);
	}

	// dwon_far
	if (p.y < height - 3) {
		flag[3] = true;
		num_valid_pixels++;
		costMin = costs[down_far];
		costMinPoint = down_far;
		for (int i = 1; i < 11; ++i) {
			if (p.y < height - 3 - 2 * i) {
				int pointTemp = down_far + 2 * i * width;
				if (costs[pointTemp] < costMin) {
					costMin = costs[pointTemp];
					costMinPoint = pointTemp;
				}
			}
		}
		down_far = costMinPoint;
		ComputeMultiViewCostVectorOld(p, plane_hypotheses[down_far], cost_array[3], helper);
	}

	// left_far
	if (p.x > 2) {
		flag[5] = true;
		num_valid_pixels++;
		costMin = costs[left_far];
		costMinPoint = left_far;
		for (int i = 1; i < 11; ++i) {
			if (p.x > 2 + 2 * i) {
				int pointTemp = left_far - 2 * i;
				if (costs[pointTemp] < costMin) {
					costMin = costs[pointTemp];
					costMinPoint = pointTemp;
				}
			}
		}
		left_far = costMinPoint;
		ComputeMultiViewCostVectorOld(p, plane_hypotheses[left_far], cost_array[5], helper);
	}

	// right_far
	if (p.x < width - 3) {
		flag[7] = true;
		num_valid_pixels++;
		costMin = costs[right_far];
		costMinPoint = right_far;
		for (int i = 1; i < 11; ++i) {
			if (p.x < width - 3 - 2 * i) {
				int pointTemp = right_far + 2 * i;
				if (costs[pointTemp] < costMin) {
					costMin = costs[pointTemp];
					costMinPoint = pointTemp;
				}
			}
		}
		right_far = costMinPoint;
		ComputeMultiViewCostVectorOld(p, plane_hypotheses[right_far], cost_array[7], helper);
	}

	// up_near
	if (p.y > 0) {
		flag[0] = true;
		num_valid_pixels++;
		costMin = costs[up_near];
		costMinPoint = up_near;
		for (int i = 0; i < 3; ++i) {
			if (p.y > 1 + i && p.x > i) {
				int pointTemp = up_near - (1 + i) * width - (i + 1);
				if (costs[pointTemp] < costMin) {
					costMin = costs[pointTemp];
					costMinPoint = pointTemp;
				}
			}
			if (p.y > 1 + i && p.x < width - 1 - i) {
				int pointTemp = up_near - (1 + i) * width + (i + 1);
				if (costs[pointTemp] < costMin) {
					costMin = costs[pointTemp];
					costMinPoint = pointTemp;
				}
			}
		}
		up_near = costMinPoint;
		ComputeMultiViewCostVectorOld(p, plane_hypotheses[up_near], cost_array[0], helper);
	}

	// down_near
	if (p.y < height - 1) {
		flag[2] = true;
		num_valid_pixels++;
		costMin = costs[down_near];
		costMinPoint = down_near;
		for (int i = 0; i < 3; ++i) {
			if (p.y < height - 2 - i && p.x > i) {
				int pointTemp = down_near + (1 + i) * width - (i + 1);
				if (costs[pointTemp] < costMin) {
					costMin = costs[pointTemp];
					costMinPoint = pointTemp;
				}
			}
			if (p.y < height - 2 - i && p.x < width - 1 - i) {
				int pointTemp = down_near + (1 + i) * width + (i + 1);
				if (costs[pointTemp] < costMin) {
					costMin = costs[pointTemp];
					costMinPoint = pointTemp;
				}
			}
		}
		down_near = costMinPoint;
		ComputeMultiViewCostVectorOld(p, plane_hypotheses[down_near], cost_array[2], helper);
	}

	// left_near
	if (p.x > 0) {
		flag[4] = true;
		num_valid_pixels++;
		costMin = costs[left_near];
		costMinPoint = left_near;
		for (int i = 0; i < 3; ++i) {
			if (p.x > 1 + i && p.y > i) {
				int pointTemp = left_near - (1 + i) - (i + 1) * width;
				if (costs[pointTemp] < costMin) {
					costMin = costs[pointTemp];
					costMinPoint = pointTemp;
				}
			}
			if (p.x > 1 + i && p.y < height - 1 - i) {
				int pointTemp = left_near - (1 + i) + (i + 1) * width;
				if (costs[pointTemp] < costMin) {
					costMin = costs[pointTemp];
					costMinPoint = pointTemp;
				}
			}
		}
		left_near = costMinPoint;
		ComputeMultiViewCostVectorOld(p, plane_hypotheses[left_near], cost_array[4], helper);
	}

	// right_near
	if (p.x < width - 1) {
		flag[6] = true;
		num_valid_pixels++;
		costMin = costs[right_near];
		costMinPoint = right_near;
		for (int i = 0; i < 3; ++i) {
			if (p.x < width - 2 - i && p.y > i) {
				int pointTemp = right_near + (1 + i) - (i + 1) * width;
				if (costs[pointTemp] < costMin) {
					costMin = costs[pointTemp];
					costMinPoint = pointTemp;
				}
			}
			if (p.x < width - 2 - i && p.y < height - 1 - i) {
				int pointTemp = right_near + (1 + i) + (i + 1) * width;
				if (costs[pointTemp] < costMin) {
					costMin = costs[pointTemp];
					costMinPoint = pointTemp;
				}
			}
		}
		right_near = costMinPoint;
		ComputeMultiViewCostVectorOld(p, plane_hypotheses[right_near], cost_array[6], helper);
	}

	const int positions[8] = { up_near, up_far, down_near, down_far, left_near, left_far, right_near, right_far };

	// Multi-hypothesis Joint View Selection
	uchar* view_weights = &(helper->view_weight_cuda[center * MAX_IMAGES]);
	for (int i = 0; i < MAX_IMAGES; ++i) {
		view_weights[i] = 0;
	}
	float view_selection_priors[32] = { 0.0f };

	int neighbor_positions[4] = { center - width, center + width, center - 1, center + 1 };
	for (int i = 0; i < 4; ++i) {
		if (flag[2 * i]) {
			for (int j = 0; j < num_images - 1; ++j) {
				if (isSet(selected_views[neighbor_positions[i]], j) == 1) {
					view_selection_priors[j] += 0.9f;
				}
				else {
					view_selection_priors[j] += 0.1f;
				}
			}
		}
	}

	float sampling_probs[32] = { 0.0f };
	float cost_threshold = 0.8 * expf((iter) * (iter) / (-90.0f));
	for (int i = 0; i < num_images - 1; i++) {
		float count = 0;
		int count_false = 0;
		float tmpw = 0;
		for (int j = 0; j < 8; j++) {
			if (cost_array[j][i] < cost_threshold) {
				tmpw += expf(cost_array[j][i] * cost_array[j][i] / (-0.18f));
				count++;
			}
			if (cost_array[j][i] > 1.2f) {
				count_false++;
			}
		}
		if (count > 2 && count_false < 3) {
			sampling_probs[i] = tmpw / count;
		}
		else if (count_false < 3) {
			sampling_probs[i] = expf(cost_threshold * cost_threshold / (-0.32f));
		}
		sampling_probs[i] = sampling_probs[i] * view_selection_priors[i];
	}

	TransformPDFToCDF(sampling_probs, num_images - 1);
	for (int sample = 0; sample < 15; ++sample) {
		const float rand_prob = hiprand_uniform(&rand_states[center]) - FLT_EPSILON;

		for (int image_id = 0; image_id < num_images - 1; ++image_id) {
			const float prob = sampling_probs[image_id];
			if (prob > rand_prob) {
				view_weights[image_id] += 1;
				break;
			}
		}
	}

	unsigned int temp_selected_views = 0;
	int num_selected_view = 0;
	float weight_norm = 0;

	for (int i = 0; i < num_images - 1; ++i) {
		if (view_weights[i] > 0) {
			setBit(&temp_selected_views, i);
			weight_norm += view_weights[i];
			num_selected_view++;
		}
	}

	float final_costs[8] = { 0.0f };

	for (int i = 0; i < 8; ++i) {
		for (int j = 0; j < num_images - 1; ++j) {
			if (view_weights[j] > 0) {
				final_costs[i] += view_weights[j] * cost_array[i][j];
			}
		}
		final_costs[i] /= weight_norm;
	}

	const int min_cost_idx = FindMinCostIndex(final_costs, 8);

	float cost_vector_now[32] = { 2.0f };
	ComputeMultiViewCostVectorOld(p, plane_hypotheses[center], cost_vector_now, helper);
	float cost_now = 0.0f;

	for (int i = 0; i < num_images - 1; ++i) {
		if (params->geom_consistency && params->use_impetus) {
			cost_now += view_weights[i] * (cost_vector_now[i] + params->geom_factor * ComputeGeomConsistencyCost(p, i + 1, plane_hypotheses[center], helper));
		}
		else {
			cost_now += view_weights[i] * cost_vector_now[i];
		}
	}
	cost_now /= weight_norm;
	costs[center] = cost_now;
	float depth_now = ComputeDepthfromPlaneHypothesis(cameras[0], plane_hypotheses[center], p);
	float4 plane_hypotheses_now = plane_hypotheses[center];

	if (flag[min_cost_idx]) {
		float depth_before = ComputeDepthfromPlaneHypothesis(cameras[0], plane_hypotheses[positions[min_cost_idx]], p);

		if (depth_before >= params->depth_min && depth_before <= params->depth_max && final_costs[min_cost_idx] < cost_now) {
			depth_now = depth_before;
			plane_hypotheses_now = plane_hypotheses[positions[min_cost_idx]];
			cost_now = final_costs[min_cost_idx];
			selected_views[center] = temp_selected_views;
		}
	}
	PlaneHypothesisRefinementStrong(&plane_hypotheses_now, &depth_now, &cost_now, &rand_states[center], view_weights, weight_norm, p, helper);

	if (params->state == REFINE_INIT) {
		if (cost_now < costs[center] - 0.1) {
			costs[center] = cost_now;
			plane_hypotheses[center] = plane_hypotheses_now;
		}
	}
	else {
		costs[center] = cost_now;
		plane_hypotheses[center] = plane_hypotheses_now;
	}
}

__device__ void CheckerboardPropagationWeak(
	const int2 p,
	const int iter,
	DataPassHelper* helper
) {
	const int width = helper->width;
	const int height = helper->height;
	float4* plane_hypotheses = helper->plane_hypotheses_cuda;
	float* costs = helper->costs_cuda;
	hiprandState* rand_states = helper->rand_states_cuda;
	unsigned int* selected_views = helper->selected_views_cuda;
	PatchMatchParams* params = helper->params;
	const Camera* cameras = helper->cameras_cuda;
	int num_images = params->num_images;

	if (p.x >= width || p.y >= height) {
		return;
	}

	const int center = p.y * width + p.x;

	// Adaptive Checkerboard Sampling
	float cost_array[8][32] = { 2.0f };
	bool flag[8] = { false };
	int num_valid_pixels = 0;

	int positions[8] = { 0 };
	float4 new_plane_hypothesis[8];

	for (int i = 0; i < 8; ++i) {
		const auto anchor_pt = GetAnchorPoint(p, i + 1, helper);
		if (anchor_pt.x == -1 || anchor_pt.y == -1 || helper->weak_info_cuda[anchor_pt.x + anchor_pt.y * width] != STRONG) {
			flag[i] = false;
			continue;
		}
		positions[i] = anchor_pt.x + anchor_pt.y * width;
		flag[i] = true;
		num_valid_pixels++;
		ComputeMultiViewCostVectorNew(p, plane_hypotheses[anchor_pt.x + anchor_pt.y * width], cost_array[i], helper);
		new_plane_hypothesis[i] = plane_hypotheses[anchor_pt.x + anchor_pt.y * width];
	}

	// Multi-hypothesis Joint View Selection
	uchar* view_weights = &(helper->view_weight_cuda[center * MAX_IMAGES]);
	for (int i = 0; i < MAX_IMAGES; ++i) {
		view_weights[i] = 0;
	}
	float view_selection_priors[32] = { 0.0f };
	for (int i = 0; i < 8; ++i) {
		const auto anchor_pt = GetAnchorPoint(p, i + 1, helper);
		if (anchor_pt.x == -1 || anchor_pt.y == -1) {
			continue;
		}
		for (int j = 0; j < num_images - 1; ++j) {
			if (isSet(selected_views[anchor_pt.x + anchor_pt.y * width], j) == 1) {
				view_selection_priors[j] += 0.9f;
			}
			else {
				view_selection_priors[j] += 0.1f;
			}
		}
	}

	float sampling_probs[32] = { 0.0f };
	float cost_threshold = 0.8 * expf((iter) * (iter) / (-90.0f));
	for (int i = 0; i < num_images - 1; i++) {
		float count = 0;
		int count_false = 0;
		float tmpw = 0;
		for (int j = 0; j < 8; j++) {
			if (cost_array[j][i] < cost_threshold) {
				tmpw += expf(cost_array[j][i] * cost_array[j][i] / (-0.18f));
				count++;
			}
			if (cost_array[j][i] > 1.2f) {
				count_false++;
			}
		}
		if (count > 2 && count_false < 3) {
			sampling_probs[i] = tmpw / count;
		}
		else if (count_false < 3) {
			sampling_probs[i] = expf(cost_threshold * cost_threshold / (-0.32f));
		}
		sampling_probs[i] = sampling_probs[i] * view_selection_priors[i];
	}

	TransformPDFToCDF(sampling_probs, num_images - 1);
	for (int sample = 0; sample < 15; ++sample) {
		const float rand_prob = hiprand_uniform(&rand_states[center]) - FLT_EPSILON;

		for (int image_id = 0; image_id < num_images - 1; ++image_id) {
			const float prob = sampling_probs[image_id];
			if (prob > rand_prob) {
				view_weights[image_id] += 1;
				break;
			}
		}
	}

	unsigned int temp_selected_views = 0;
	int num_selected_view = 0;
	float weight_norm = 0;

	for (int i = 0; i < num_images - 1; ++i) {
		if (view_weights[i] > 0) {
			setBit(&temp_selected_views, i);
			weight_norm += view_weights[i];
			num_selected_view++;
		}
	}

	float final_costs[8] = { 0.0f };

	for (int i = 0; i < 8; ++i) {
		for (int j = 0; j < num_images - 1; ++j) {
			if (view_weights[j] > 0) {
				if (params->geom_consistency) {
					if (flag[i]) {
						final_costs[i] += view_weights[j] * (cost_array[i][j] + params->geom_factor * ComputeGeomConsistencyCost(p, j + 1, plane_hypotheses[positions[i]], helper));
					}
					else {
						final_costs[i] += view_weights[j] * (cost_array[i][j] + params->geom_factor * 3.0f);
					}
				}
				else {
					final_costs[i] += view_weights[j] * cost_array[i][j];
				}
			}
		}
		final_costs[i] /= weight_norm;
	}

	const int min_cost_idx = FindMinCostIndex(final_costs, 8);

	float cost_vector_now[32] = { 2.0f };
	ComputeMultiViewCostVectorNew(p, plane_hypotheses[center], cost_vector_now, helper);
	float cost_now = 0.0f;

	for (int i = 0; i < num_images - 1; ++i) {
		if (params->geom_consistency) {
			cost_now += view_weights[i] * (cost_vector_now[i] + params->geom_factor * ComputeGeomConsistencyCost(p, i + 1, plane_hypotheses[center], helper));
		}
		else {
			cost_now += view_weights[i] * cost_vector_now[i];
		}
	}
	cost_now /= weight_norm;
	costs[center] = cost_now;
	float depth_now = ComputeDepthfromPlaneHypothesis(cameras[0], plane_hypotheses[center], p);
	float4 plane_hypotheses_now = plane_hypotheses[center];

	if (flag[min_cost_idx]) {
		float depth_before = ComputeDepthfromPlaneHypothesis(cameras[0], new_plane_hypothesis[min_cost_idx], p);
		if (depth_before >= params->depth_min && depth_before <= params->depth_max && final_costs[min_cost_idx] < cost_now) {
			depth_now = depth_before;
			plane_hypotheses_now = new_plane_hypothesis[min_cost_idx];
			cost_now = final_costs[min_cost_idx];
			selected_views[center] = temp_selected_views;
		}
	}
	PlaneHypothesisRefinementWeak(&plane_hypotheses_now, &depth_now, &cost_now, &rand_states[center], view_weights, weight_norm, p, helper);

	if (params->state == REFINE_INIT) {
		if (cost_now < costs[center] - 0.1) {
			costs[center] = cost_now;
			plane_hypotheses[center] = plane_hypotheses_now;
		}
	}
	else {
		costs[center] = cost_now;
		plane_hypotheses[center] = plane_hypotheses_now;
	}
}

__global__ void BlackPixelUpdateWeak(const int iter, DataPassHelper* helper)
{
	int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);

	if (threadIdx.x % 2 == 0) {
		p.y = p.y * 2;
	}
	else {
		p.y = p.y * 2 + 1;
	}
	if (p.x >= helper->width || p.y >= helper->height) {
		return;
	}

	if (helper->weak_info_cuda[p.x + p.y * helper->width] == WEAK) {
		CheckerboardPropagationWeak(p, iter, helper);
	}
}

__global__ void RedPixelUpdateWeak(const int iter, DataPassHelper* helper)
{
	int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);

	if (threadIdx.x % 2 == 0) {
		p.y = p.y * 2 + 1;
	}
	else {
		p.y = p.y * 2;
	}
	if (p.x >= helper->width || p.y >= helper->height) {
		return;
	}
	if (helper->weak_info_cuda[p.x + p.y * helper->width] == WEAK) {
		CheckerboardPropagationWeak(p, iter, helper);
	}
}

__global__ void BlackPixelUpdateStrong(const int iter, DataPassHelper* helper)
{
	int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);

	if (threadIdx.x % 2 == 0) {
		p.y = p.y * 2;
	}
	else {
		p.y = p.y * 2 + 1;
	}
	if (p.x >= helper->width || p.y >= helper->height) {
		return;
	}
	if (helper->weak_info_cuda[p.x + p.y * helper->width] == WEAK) {
		return;
	}

	CheckerboardPropagationStrong(p, iter, helper);
}

__global__ void RedPixelUpdateStrong(const int iter, DataPassHelper* helper)
{
	int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);

	if (threadIdx.x % 2 == 0) {
		p.y = p.y * 2 + 1;
	}
	else {
		p.y = p.y * 2;
	}
	if (p.x >= helper->width || p.y >= helper->height) {
		return;
	}
	if (helper->weak_info_cuda[p.x + p.y * helper->width] == WEAK) {
		return;
	}

	CheckerboardPropagationStrong(p, iter, helper);
}

__global__ void GetDepthandNormal(
	DataPassHelper* helper
) {
	const int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
	Camera* cameras = helper->cameras_cuda;
	float4* plane_hypotheses = helper->plane_hypotheses_cuda;
	const int width = helper->width;;
	const int height = helper->height;

	if (p.x >= width || p.y >= height) {
		return;
	}
	const int center = p.y * width + p.x;
	plane_hypotheses[center].w = ComputeDepthfromPlaneHypothesis(cameras[0], plane_hypotheses[center], p);
	plane_hypotheses[center] = TransformNormal(cameras[0], plane_hypotheses[center]);
}

__device__ void CheckerboardFilterStrong(
	const int2 p,
	DataPassHelper* helper
) {
	int width = helper->width;
	int height = helper->height;
	if (p.x >= width || p.y >= height) {
		return;
	}
	float4* plane_hypotheses = helper->plane_hypotheses_cuda;
	float* costs = helper->costs_cuda;
	const int center = p.y * width + p.x;

	float filter[21];
	int index = 0;

	filter[index++] = plane_hypotheses[center].w;

	// Left
	const int left = center - 1;
	const int leftleft = center - 3;

	// Up
	const int up = center - width;
	const int upup = center - 3 * width;

	// Down
	const int down = center + width;
	const int downdown = center + 3 * width;

	// Right
	const int right = center + 1;
	const int rightright = center + 3;

	if (costs[center] < 0.001f) {
		return;
	}

	if (p.y > 0 && helper->weak_info_cuda[up] == STRONG) {
		filter[index++] = plane_hypotheses[up].w;
	}
	if (p.y > 2 && helper->weak_info_cuda[upup] == STRONG) {
		filter[index++] = plane_hypotheses[upup].w;
	}
	if (p.y > 4 && helper->weak_info_cuda[upup - width * 2] == STRONG) {
		filter[index++] = plane_hypotheses[upup - width * 2].w;
	}
	if (p.y < height - 1 && helper->weak_info_cuda[down] == STRONG) {
		filter[index++] = plane_hypotheses[down].w;
	}
	if (p.y < height - 3 && helper->weak_info_cuda[downdown] == STRONG) {
		filter[index++] = plane_hypotheses[downdown].w;
	}
	if (p.y < height - 5 && helper->weak_info_cuda[downdown + width * 2] == STRONG) {
		filter[index++] = plane_hypotheses[downdown + width * 2].w;
	}
	if (p.x > 0 && helper->weak_info_cuda[left] == STRONG) {
		filter[index++] = plane_hypotheses[left].w;
	}
	if (p.x > 2 && helper->weak_info_cuda[leftleft] == STRONG) {
		filter[index++] = plane_hypotheses[leftleft].w;
	}
	if (p.x > 4 && helper->weak_info_cuda[leftleft - 2] == STRONG) {
		filter[index++] = plane_hypotheses[leftleft - 2].w;
	}
	if (p.x < width - 1 && helper->weak_info_cuda[right] == STRONG) {
		filter[index++] = plane_hypotheses[right].w;
	}
	if (p.x < width - 3 && helper->weak_info_cuda[rightright] == STRONG) {
		filter[index++] = plane_hypotheses[rightright].w;
	}
	if (p.x < width - 5 && helper->weak_info_cuda[rightright + 2] == STRONG) {
		filter[index++] = plane_hypotheses[rightright + 2].w;
	}
	if (p.y > 0 && p.x < width - 2 && helper->weak_info_cuda[up + 2] == STRONG) {
		filter[index++] = plane_hypotheses[up + 2].w;
	}
	if (p.y < height - 1 && p.x < width - 2 && helper->weak_info_cuda[down + 2] == STRONG) {
		filter[index++] = plane_hypotheses[down + 2].w;
	}
	if (p.y > 0 && p.x > 1 && helper->weak_info_cuda[up - 2] == STRONG)
	{
		filter[index++] = plane_hypotheses[up - 2].w;
	}
	if (p.y < height - 1 && p.x>1 && helper->weak_info_cuda[down - 2] == STRONG) {
		filter[index++] = plane_hypotheses[down - 2].w;
	}
	if (p.x > 0 && p.y > 2 && helper->weak_info_cuda[left - width * 2] == STRONG)
	{
		filter[index++] = plane_hypotheses[left - width * 2].w;
	}
	if (p.x < width - 1 && p.y>2 && helper->weak_info_cuda[right - width * 2] == STRONG)
	{
		filter[index++] = plane_hypotheses[right - width * 2].w;
	}
	if (p.x > 0 && p.y < height - 2 && helper->weak_info_cuda[left + width * 2] == STRONG) {
		filter[index++] = plane_hypotheses[left + width * 2].w;
	}
	if (p.x < width - 1 && p.y < height - 2 && helper->weak_info_cuda[right + width * 2] == STRONG) {
		filter[index++] = plane_hypotheses[right + width * 2].w;
	}

	sort_small(filter, index);
	int median_index = index / 2;
	if (index % 2 == 0) {
		plane_hypotheses[center].w = (filter[median_index - 1] + filter[median_index]) / 2;
	}
	else {
		plane_hypotheses[center].w = filter[median_index];
	}
}

__global__ void BlackPixelFilterStrong(DataPassHelper* helper)
{
	int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
	if (threadIdx.x % 2 == 0) {
		p.y = p.y * 2;
	}
	else {
		p.y = p.y * 2 + 1;
	}
	if (p.x >= helper->width || p.y >= helper->height) {
		return;
	}
	if (helper->weak_info_cuda[p.x + p.y * helper->width] != WEAK) {
		CheckerboardFilterStrong(p, helper);
	}
}

__global__ void RedPixelFilterStrong(DataPassHelper* helper)
{
	int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
	if (threadIdx.x % 2 == 0) {
		p.y = p.y * 2 + 1;
	}
	else {
		p.y = p.y * 2;
	}
	if (p.x >= helper->width || p.y >= helper->height) {
		return;
	}
	if (helper->weak_info_cuda[p.x + p.y * helper->width] != WEAK) {
		CheckerboardFilterStrong(p, helper);
	}
}

__global__ void GenAnchors(
	DataPassHelper* helper
) {
	int width = helper->width;
	int height = helper->height;
	const int2 point = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
	if (point.x >= width || point.y >= height) {
		return;
	}
	const unsigned center = point.x + point.y * width;
	const uchar* weak_info = helper->weak_info_cuda;
	if (weak_info[center] != WEAK) {
		return;
	}
	const int min_margin = 6;
	const float depth_diff = helper->params->depth_max - helper->params->depth_min;
	const int* anchors_map = helper->anchors_map_cuda;
	const PatchMatchParams* params = helper->params;
	const short2* weak_nearest_strong = helper->weak_nearest_strong;
	const Camera& camera = helper->cameras_cuda[0];
	const unsigned offset = anchors_map[center] * ANCHOR_NUM;
	const float4* plane_hypotheses = helper->plane_hypotheses_cuda;
	hiprandState* rand_state = &(helper->rand_states_cuda[center]);
	short2* anchors = &(helper->anchors_cuda[offset]);
	uchar* weak_reliable = &(helper->weak_reliable_cuda[center]);
	// init for invalid points
	for (int i = 0; i < ANCHOR_NUM; ++i) {
		anchors[i].x = -1;
		anchors[i].y = -1;
	}
	anchors[0] = make_short2(point.x, point.y); // the first point is the center point
	short2 strong_points[8 * 4];
	bool dir_valid[8 * 4];
	for (int i = 0; i < 32; ++i) {
		strong_points[i] = make_short2(-1, -1);
		dir_valid[i] = false;
	}
	int origin_direction_index = -1;
	int strong_point_size = 0;
	const int rotate_time = params->rotate_time; // max is 4 from [1, 2, 4]
	const float angle = 45.0f / rotate_time;
	const float cos_angle = cos(angle * M_PI / 180.f);
	const float sin_angle = sin(angle * M_PI / 180.f);
	const float threshhold = cos((angle / 2.0f) * M_PI / 180.0f);
	const int shift_range = MAX((int)(tan((angle / 2.0f) * M_PI / 180.0f) * 20), 1);
	const float ransac_threshold = params->ransac_threshold;

	for (int origin_direction_x = -1; origin_direction_x <= 1; ++origin_direction_x) {
		for (int origin_direction_y = -1; origin_direction_y <= 1; ++origin_direction_y) {
			if (origin_direction_x == 0 && origin_direction_y == 0) {
				continue;
			}
			float2 origin_direction = make_float2(origin_direction_x, origin_direction_y);
			NormalizeVec2(&origin_direction);
			origin_direction_index++;
			for (int rotate_iter = 0; rotate_iter < rotate_time; ++rotate_iter) {
				int dir_index = origin_direction_index * 4 + rotate_iter;
                bool stop_expand = false;
				for (int radius = 2; radius <= MAX_SEARCH_RADIUS; radius = MIN(radius * 2, radius + 25)) {
					float2 test_pt = make_float2(point.x + origin_direction.x * radius, point.y + origin_direction.y * radius);
					if (test_pt.x < 0 || test_pt.y < 0 || test_pt.x >= width || test_pt.y >= height) {
						break;
					}
					for (int radius_iter = 0; radius_iter < 4; ++radius_iter) {
						int rand_x_shift = (hiprand(rand_state) % 2 == 0 ? 1 : -1) * hiprand(rand_state) % shift_range;
						int rand_y_shift = (hiprand(rand_state) % 2 == 0 ? 1 : -1) * hiprand(rand_state) % shift_range;
						float2 direction = make_float2(origin_direction.x * 20 + rand_x_shift, origin_direction.y * 20 + rand_y_shift);
						NormalizeVec2(&direction);
						short2 anchor_pt = make_short2(point.x + direction.x * radius, point.y + direction.y * radius);
						if (anchor_pt.x < min_margin || anchor_pt.y < min_margin || anchor_pt.x >= width - min_margin || anchor_pt.y >= height - min_margin) {
							continue;
						}
						int anchor_pt_center = anchor_pt.x + anchor_pt.y * width;
                        anchor_pt = weak_nearest_strong[anchor_pt_center];
                        if (anchor_pt.x == -1 || anchor_pt.y == -1) {
                            continue;
                        }
                        anchor_pt_center = anchor_pt.x + anchor_pt.y * width;

						float2 test_direction = make_float2(anchor_pt.x - point.x, anchor_pt.y - point.y);
						NormalizeVec2(&test_direction);
						float cos_angle = Vec2DotVec2(test_direction, origin_direction);
						if (cos_angle > threshhold) {
							strong_points[dir_index] = anchor_pt;
							dir_valid[dir_index] = true;
							strong_point_size++;
							break;
						}
					}
					if (dir_valid[dir_index]) {
						break;
					}
                    if (stop_expand) {
                        break;
                    }
				}
				// rotate
				{
					float2 rotated_direction;
					rotated_direction.x = origin_direction.x * cos_angle - origin_direction.y * sin_angle;
					rotated_direction.y = origin_direction.x * sin_angle + origin_direction.y * cos_angle;
					NormalizeVec2(&rotated_direction);
					origin_direction = rotated_direction;
				}
			}
		}
	}

	if (strong_point_size <= 3) {
		*weak_reliable = 0;
		return;
	}
	float4 best_plane;
	int use_a_index = -1, use_b_index = -1, use_c_index = -1;
	bool has_valid_plane = false;
	short2 strong_points_valid[8 * 4];
	float3 strong_points_valid_3d[8 * 4];
	int valid_count = 0;
	float X[3];
	Get3DPoint(camera, point, plane_hypotheses[center].w, X);
	float3 center_point_world = make_float3(X[0], X[1], X[2]);
	for (int i = 0; i < 32; ++i) {
		strong_points_valid[i] = make_short2(-1, -1);
		if (dir_valid[i]) {
			const auto& strong_point = strong_points[i];
			int strong_point_center = strong_point.x + strong_point.y * width;
			strong_points_valid[valid_count] = strong_points[i];
			Get3DPoint(camera, strong_point, plane_hypotheses[strong_point_center].w, X);
			strong_points_valid_3d[valid_count] = make_float3(X[0], X[1], X[2]);
			valid_count++;
		}
	}
	{	// RANSAC to find a good plane
		int iteration = 50;
		float min_cost = FLT_MAX;
		int max_count = 3;
		while (iteration--) {
			int a_index = hiprand(rand_state) % valid_count;
			int b_index = hiprand(rand_state) % valid_count;
			int c_index = hiprand(rand_state) % valid_count;
			if (a_index == b_index || b_index == c_index || a_index == c_index) {
				continue;
			}
			// compute triangle
			if (!PointinTriangle(strong_points_valid[a_index], strong_points_valid[b_index], strong_points_valid[c_index], point)) {
				continue;
			}
			const float3& A = strong_points_valid_3d[a_index];
			const float3& B = strong_points_valid_3d[b_index];
			const float3& C = strong_points_valid_3d[c_index];
			float3 A_C = make_float3(A.x - C.x, A.y - C.y, A.z - C.z);
			float3 B_C = make_float3(B.x - C.x, B.y - C.y, B.z - C.z);
			float4 cross_vec;
			cross_vec.x = A_C.y * B_C.z - B_C.y * A_C.z;
			cross_vec.y = -(A_C.x * B_C.z - B_C.x * A_C.z);
			cross_vec.z = A_C.x * B_C.y - B_C.x * A_C.y;
			if ((cross_vec.x == 0 && cross_vec.y == 0 && cross_vec.z == 0) || isnan(cross_vec.x) || isnan(cross_vec.y) || isnan(cross_vec.z)) {
				continue;
			}
			NormalizeVec3(&cross_vec);
			cross_vec.w = -(cross_vec.x * A.x + cross_vec.y * A.y + cross_vec.z * A.z);
			int temp_count = 0;
			float strong_dist = 0.0f;
			for (int strong_index = 0; strong_index < valid_count; ++strong_index) {
				const float3& temp_point = strong_points_valid_3d[strong_index];
				float distance = fabs(cross_vec.x * temp_point.x + cross_vec.y * temp_point.y + cross_vec.z * temp_point.z + cross_vec.w);
				if (distance / depth_diff < ransac_threshold) {
					temp_count++;
					strong_dist += distance;
				}
			}
			if (temp_count < 6) {
				continue;
			}
			if (temp_count > max_count) {
				max_count = temp_count;
				const float center_distance = fabs(cross_vec.x * center_point_world.x + cross_vec.y * center_point_world.y + cross_vec.z * center_point_world.z + cross_vec.w);
				strong_dist /= temp_count;
				min_cost = center_distance;
				best_plane = cross_vec;
				has_valid_plane = true;
				use_a_index = a_index;
				use_b_index = b_index;
				use_c_index = c_index;
			}
			else if (temp_count == max_count) {
				const float center_distance = fabs(cross_vec.x * center_point_world.x + cross_vec.y * center_point_world.y + cross_vec.z * center_point_world.z + cross_vec.w);
				if (center_distance < min_cost) {
					max_count = temp_count;
					strong_dist /= temp_count;
					min_cost = center_distance;
					best_plane = cross_vec;
					use_a_index = a_index;
					use_b_index = b_index;
					use_c_index = c_index;
				}
			}
		}
	}
	if (!has_valid_plane) {
		*weak_reliable = 0;
		return;
	}
	float weight[32];
	for (int i = 0; i < valid_count; ++i) {
		const float3& temp_point = strong_points_valid_3d[i];
		float distance = fabs(best_plane.x * temp_point.x + best_plane.y * temp_point.y + best_plane.z * temp_point.z + best_plane.w);
		if (distance / depth_diff >= ransac_threshold) {
			strong_points_valid[i] = make_short2(-1, -1);
			weight[i] = FLT_MAX;
			continue;
		}
		if (i == use_a_index || i == use_b_index || i == use_c_index) {
			distance -= 1;
		}
		weight[i] = distance;
	}
	sort_small_weighted(strong_points_valid, weight, valid_count);
	for (int i = 1; i < ANCHOR_NUM; ++i) {
		anchors[i] = strong_points_valid[i - 1];
        if (anchors[i].x == -1 || anchors[i].y == -1) {
            continue;
        }
	}
	*weak_reliable = 1;
}

__global__ void NeigbourUpdate(
	DataPassHelper* helper
) {
	const int2 point = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
	const int width = helper->width;
	const int height = helper->height;
	if (point.x >= width || point.y >= height) {
		return;
	}
	const int center = point.x + point.y * width;
	if (helper->weak_info_cuda[center] != WEAK) {
		return;
	}
	if (helper->weak_reliable_cuda[center] != 1) {
		helper->weak_info_cuda[center] = UNKNOWN;
	}
}


__global__ void DepthToWeak(DataPassHelper *helper) {
	const int2 point = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
	const int width = helper->width;
	const int height = helper->height;
	if (point.x >= width || point.y >= height) {
		return;
	}

	const int min_margin = 6;
	const int center = point.x + point.y * width;

	if (point.x < min_margin || point.y < min_margin || point.x >= width - min_margin || point.y >= height - min_margin) {
		helper->weak_info_cuda[center] = UNKNOWN;
		return;
	}
	const auto &image = helper->texture_objects_cuda[0].images[0];
	const float center_pix = tex2D<float>(image, point.x + 0.5f, point.y + 0.5f);

	const Camera *cameras = helper->cameras_cuda;
	const unsigned *selected_views = helper->selected_views_cuda;
	const int num_images = helper->params->num_images;
	const uchar *view_weight = &(helper->view_weight_cuda[MAX_IMAGES * center]);
	float4 origin_plane_hypothesis;
	origin_plane_hypothesis = helper->plane_hypotheses_cuda[center];
	origin_plane_hypothesis = TransformNormal2RefCam(cameras[0], origin_plane_hypothesis);
	float origin_depth = origin_plane_hypothesis.w;
	if (origin_depth == 0) {
		helper->weak_info_cuda[center] = UNKNOWN;
		return;
	}
	// compute cost now and baseline

	float base_line = 0;
	int valid_src = 0;
	float weight_normal = 0.0f;
	for (int src_index = 1; src_index < num_images; ++src_index) {
		int view_index = src_index - 1;
		if (isSet(selected_views[center], view_index)) {
			weight_normal += view_weight[view_index];
			float c_dist[3];
			c_dist[0] = cameras[0].c[0] - cameras[src_index].c[0];
			c_dist[1] = cameras[0].c[1] - cameras[src_index].c[1];
			c_dist[2] = cameras[0].c[2] - cameras[src_index].c[2];
			double temp_val = c_dist[0] * c_dist[0] + c_dist[1] * c_dist[1] + c_dist[2] * c_dist[2];
			base_line += sqrtf(temp_val);
			valid_src++;
		}
	}
	if (valid_src == 0) {
		helper->weak_info_cuda[center] = UNKNOWN;
		return;
	}
	base_line /= valid_src;

	float disp = cameras[0].K[0] * base_line / origin_depth;
	const int radius = 30;
	const int p_costs_size = 2 * radius + 1;
	float p_costs[p_costs_size];
	int increment = 1;

	for (int p_disp = -radius * increment; p_disp <= radius * increment; p_disp += increment) {

		float p_depth = cameras[0].K[0] * base_line / (disp + p_disp);
		if (p_depth < helper->params->depth_min || p_depth > helper->params->depth_max) {
			p_costs[p_disp + radius] = 2.0f;
			continue;
		}
		float4 temp_plane_hypothesis = origin_plane_hypothesis;
		temp_plane_hypothesis.w = GetDistance2Origin(cameras[0], point, p_depth, temp_plane_hypothesis);
		float p_cost = 0.0f;
		for (int src_index = 1; src_index < num_images; ++src_index) {
			int view_index = src_index - 1;
			float temp_cost = 0.0f;
 			if (isSet(selected_views[center], view_index)) {
				temp_cost += ComputeBilateralNCCOld(point, src_index, temp_plane_hypothesis, helper);
				if (helper->params->geom_consistency) {
					temp_cost += helper->params->geom_factor * ComputeGeomConsistencyCost(point, src_index, temp_plane_hypothesis, helper);
				}
				p_cost += (temp_cost * view_weight[view_index]);
			}
		}
		p_cost /= weight_normal;
		p_costs[p_disp + radius] = MIN(2.0f, p_cost);
	}
	// find peaks
	bool is_peak[p_costs_size];
	for (int i = 0; i < p_costs_size; ++i) {
		is_peak[i] = false;
	}

	int peak_count = 0;
	int min_peak = 0;
	float min_cost = 2.0f;
	for (int i = 2; i < p_costs_size - 2; ++i) {
		if (p_costs[i - 1] > p_costs[i] && p_costs[i + 1] > p_costs[i]) {
			is_peak[i] = true;
			peak_count++;
			if (p_costs[i] < min_cost) {
				min_peak = i;
				min_cost = p_costs[i];
			}
		}
	}

	if (abs(min_peak - radius) > helper->params->weak_peak_radius || p_costs[min_peak] > 0.5f) {
		helper->weak_info_cuda[center] = WEAK;
		return;
	}

	if (peak_count == 1) {
		if (p_costs[min_peak] <= 0.15f) {
			helper->weak_info_cuda[center] = STRONG;
		}
		else {
			helper->weak_info_cuda[center] = WEAK;
		}
		return;
	}

	float var = 0.0f;
	for (int i = 2; i < p_costs_size - 2; ++i) {
		if (is_peak[i] && i != min_peak) {
			float dist = p_costs[i] - min_cost;
			var += dist * dist;
		}
	}
	var = sqrtf(var);
	var /= (peak_count - 1);

	if (var > 0.2f) {
		helper->weak_info_cuda[center] = STRONG;
	} else {
		helper->weak_info_cuda[center] = WEAK;
	}
}

__global__ void WeakFilter(DataPassHelper *helper, uchar *weak_info_copy) {
    const int2 point = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    const int width = helper->width;
    const int height = helper->height;
    if (point.x >= width || point.y >= height) {
        return;
    }
    const int center = point.x + point.y * width;
    if (helper->weak_info_cuda[center] != STRONG) {
        return;
    }
    const int radius = 2;
    for (int x = - radius; x <= radius; ++x) {
        for (int y = - radius; y <= radius; ++y) {
            if (x == 0 && y == 0) {
                continue;
            }
            int2 neighbor = make_int2(point.x + x, point.y + y);
            if (neighbor.x < 0 || neighbor.x >= width || neighbor.y < 0 || neighbor.y >= height) {
                continue;
            }
            int neighbor_center = neighbor.x + neighbor.y * width;
            if (helper->weak_info_cuda[neighbor_center] == STRONG) {
                return;
            }
        }
    }
    weak_info_copy[center] = UNKNOWN;
}

__global__ void ConfidenceCompute(DataPassHelper *helper) {
    const int2 point = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    const int width = helper->width;
    const int height = helper->height;
    if (point.x >= width || point.y >= height) {
        return;
    }
    const int center = point.x + point.y * width;
    uchar *confidence = helper->confidence_cuda;
    confidence[center] = 0;

    const Camera& ref_camera = helper->cameras_cuda[0];
    const auto *params = helper->params;
    const auto selected_view = helper->selected_views_cuda[center];
    const auto ref_depth = helper->plane_hypotheses_cuda[center].w;

    if (ref_depth <= 0.0f) {
        helper->weak_info_cuda[center] = UNKNOWN;
        return;
    }
    const float3 forward_point = Get3DPointonWorld_cu(point.x, point.y, ref_depth, ref_camera);
    int num_consistence = 1; // init with 1
    const int exist_in_src_weight = 1;
    const int reproj_pixel_weight = 2;
    const int reproj_depth_weight = 2;

    for (int i = 0; i < params->num_images - 1; ++i) {
        if (!isSet(selected_view, i)) {
            continue;
        }

        int src_idx = i + 1;
        const Camera& src_camera = helper->cameras_cuda[src_idx];
        const hipTextureObject_t depth_image = helper->texture_depths_cuda[0].images[src_idx];
        float2 src_pt;
        float src_d;
        ProjectonCamera_cu(forward_point, src_camera, src_pt, src_d);
        const auto src_depth = tex2D<float>(depth_image, (int)src_pt.x + 0.5f, (int)src_pt.y + 0.5f);
        if (src_depth <= 0.0f) {
            continue;
        }

        num_consistence += exist_in_src_weight;
        float3 src_3D_pt = Get3DPointonWorld_cu(src_pt.x, src_pt.y, src_depth, src_camera);
        float2 backward_point;
        float ref_d;
        ProjectonCamera_cu(src_3D_pt, ref_camera, backward_point, ref_d);
        const float diff_col = point.x - backward_point.x;
        const float diff_row = point.y - backward_point.y;
        const float pixel_diff = sqrtf(diff_col * diff_col + diff_row * diff_row);
        if (pixel_diff <= 2.0f) {
            num_consistence += reproj_pixel_weight;
        }
        const float relative_depth_diff = fabsf(ref_depth - ref_d) / ref_depth;
        if (relative_depth_diff <= 0.02f) {
            num_consistence += reproj_depth_weight;
        }
    }
    if (num_consistence > 255) {
        num_consistence = 255;
    }
    confidence[center] = (uchar)num_consistence;
}

__global__ void LocalRefine(DataPassHelper* helper) {
	const int2 point = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
	const int width = helper->width;
	const int height = helper->height;
	if (point.x >= width || point.y >= height) {
		return;
	}

	const int center = point.x + point.y * width;

	const Camera* cameras = helper->cameras_cuda;
	const unsigned* selected_views = helper->selected_views_cuda;
	const int num_images = helper->params->num_images;
	const uchar* view_weight = &(helper->view_weight_cuda[MAX_IMAGES * center]);
	float4 origin_plane_hypothesis;
	origin_plane_hypothesis = helper->plane_hypotheses_cuda[center];
	origin_plane_hypothesis = TransformNormal2RefCam(cameras[0], origin_plane_hypothesis);
	float origin_depth = origin_plane_hypothesis.w;
	if (origin_depth == 0) {
		return;
	}

	// compute cost now and baseline
	float cost_now = 0.0f;
	float base_line = 0;
	int valid_src = 0;
	float weight_normal = 0.0f;
	for (int src_index = 1; src_index < num_images; ++src_index) {
		int view_index = src_index - 1;
		if (isSet(selected_views[center], view_index)) {
			float4 temp_plane_hypothesis = origin_plane_hypothesis;
			temp_plane_hypothesis.w = GetDistance2Origin(cameras[0], point, origin_depth, temp_plane_hypothesis);
			float temp_cost = ComputeBilateralNCCOld(point, src_index, temp_plane_hypothesis, helper);
			if (helper->params->geom_consistency) {
				temp_cost += helper->params->geom_factor * ComputeGeomConsistencyCost(point, src_index, temp_plane_hypothesis, helper);
			}
			cost_now += (temp_cost * view_weight[view_index]);
			weight_normal += view_weight[view_index];
			float c_dist[3];
			c_dist[0] = cameras[0].c[0] - cameras[src_index].c[0];
			c_dist[1] = cameras[0].c[1] - cameras[src_index].c[1];
			c_dist[2] = cameras[0].c[2] - cameras[src_index].c[2];
			double temp_val = c_dist[0] * c_dist[0] + c_dist[1] * c_dist[1] + c_dist[2] * c_dist[2];
			base_line += sqrtf(temp_val);
			valid_src++;
		}
	}

	if (weight_normal == 0 || valid_src == 0) {
		return;
	}

	cost_now /= weight_normal;
	base_line /= valid_src;

	float disp = cameras[0].K[0] * base_line / origin_depth;
	const int radius = 5;

	float min_cost = 2.0f;
	float best_depth = origin_depth;
	for (int p_disp = -radius; p_disp <= radius; ++p_disp) {
		float p_depth = cameras[0].K[0] * base_line / (disp + p_disp);
		if (p_depth < helper->params->depth_min || p_depth > helper->params->depth_max) {
			continue;
		}
		float4 temp_plane_hypothesis = origin_plane_hypothesis;
		temp_plane_hypothesis.w = GetDistance2Origin(cameras[0], point, p_depth, temp_plane_hypothesis);
		float temp_cost = 0.0f;
		for (int src_index = 1; src_index < num_images; ++src_index) {
			int view_index = src_index - 1;
			if (isSet(selected_views[center], view_index)) {
				temp_cost += (ComputeBilateralNCCOld(point, src_index, temp_plane_hypothesis, helper) * view_weight[view_index]);
				if (helper->params->geom_consistency) {
					temp_cost += (helper->params->geom_factor * ComputeGeomConsistencyCost(point, src_index, temp_plane_hypothesis, helper) * view_weight[view_index]);
				}
			}
		}
		temp_cost /= weight_normal;
		if (temp_cost < min_cost) {
			min_cost = temp_cost;
			best_depth = p_depth;
		}
	}
	if (cost_now - min_cost > 0.1) {
		helper->plane_hypotheses_cuda[center].w = best_depth;
	}
}

__global__ void FindNearestStrongPoint(DataPassHelper* helper) {
    const int2 point = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    const int width = helper->width;
    const int height = helper->height;
    if (point.x >= width || point.y >= height) {
        return;
    }
    const uchar* weak_info = helper->weak_info_cuda;
    const uchar* confidence = helper->confidence_cuda;
    short2* weak_nearest_strong = helper->weak_nearest_strong;
    const int center = point.x + point.y * width;
    weak_nearest_strong[center].x = -1;
    weak_nearest_strong[center].y = -1;
    const uchar center_confidence = confidence[center];

    if (weak_info[center] == WEAK || weak_info[center] == UNKNOWN) {
        uchar best_confidence = 0;
        short2 best_point = make_short2(-1, -1);
        float min_dist = FLT_MAX;
        const int radius = 100;
        for (int x = -radius; x <= radius; ++x) {
            for (int y = -radius; y <= radius; ++y) {
                short2 temp_point = make_short2(point.x + x, point.y + y);
                if (temp_point.x < 0 || temp_point.x >= width || temp_point.y < 0 || temp_point.y >= height) {
                    continue;
                }
                int temp_center = temp_point.x + temp_point.y * width;
                if (weak_info[temp_center] != STRONG) {
                    continue;
                }
                if (confidence[temp_center] < center_confidence) {
                    continue;
                }
                float temp_dist = sqrtf(x * x + y * y);
                if (temp_dist < min_dist) {
                    min_dist = temp_dist;
                    best_point = temp_point;
                    best_confidence = confidence[temp_center];
                } else if (temp_dist == min_dist) {
                    if (confidence[temp_center] > best_confidence) {
                        best_point = temp_point;
                        best_confidence = confidence[temp_center];
                    }
                }
            }
        }
        weak_nearest_strong[center] = best_point;
    } else if (weak_info[center] == STRONG) {
        weak_nearest_strong[center] = make_short2(point.x, point.y);
    }
}

__global__ void RANSACToGetFitPlane(DataPassHelper* helper) {
	const int2 point = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
	const int width = helper->width;
	const int height = helper->height;
	if (point.x >= width || point.y >= height) {
		return;
	}
	const uchar* weak_info = helper->weak_info_cuda;
	const int center = point.x + point.y * width;
	float4* plane_hypotheses = helper->plane_hypotheses_cuda;
	float4* fit_plane_hypothese = helper->fit_plane_hypotheses_cuda;
	if (weak_info[center] != WEAK) {
		fit_plane_hypothese[center] = plane_hypotheses[center];
		return;
	}
	// make sure that the plane is in the ref camera coord
	hiprandState* rand_state = &(helper->rand_states_cuda[center]);
	const auto& camera = helper->cameras_cuda[0];

	short2 strong_points[ANCHOR_NUM - 1];
	float3 strong_points_3d[ANCHOR_NUM - 1];
	int strong_count = 0;
	float X[3];
	for (int i = 1; i < ANCHOR_NUM; ++i) {
		short2 temp_point = GetAnchorPoint(point, i, helper);
		if (temp_point.x == -1 || temp_point.y == -1) {
			continue;
		}
		strong_points[strong_count].x = temp_point.x;
		strong_points[strong_count].y = temp_point.y;
		// get 3d point in ref camera coord
		const int temp_center = temp_point.x + temp_point.y * width;
		float depth = ComputeDepthfromPlaneHypothesis(camera, plane_hypotheses[temp_center], make_int2(temp_point.x, temp_point.y));
		Get3DPoint(camera, strong_points[strong_count], depth, X);
		strong_points_3d[strong_count].x = X[0];
		strong_points_3d[strong_count].y = X[1];
		strong_points_3d[strong_count].z = X[2];
		strong_count++;
	}
	if (strong_count < 3) {
		fit_plane_hypothese[center] = plane_hypotheses[center];
		return;
	}

	int iteration = 50;
	float min_cost = FLT_MAX;
	float4 best_plane;
	bool has_best_plane = false;
	while (iteration--)
	{
		int a_index = hiprand(rand_state) % strong_count;
		int b_index = hiprand(rand_state) % strong_count;
		int c_index = hiprand(rand_state) % strong_count;

		if (a_index == b_index || b_index == c_index || a_index == c_index) {
			continue;
		}

		if (!PointinTriangle(strong_points[a_index], strong_points[b_index], strong_points[c_index], point)) {
			continue;
		}

		const float3& A = strong_points_3d[a_index];
		const float3& B = strong_points_3d[b_index];
		const float3& C = strong_points_3d[c_index];

		float3 A_C = make_float3(A.x - C.x, A.y - C.y, A.z - C.z);
		float3 B_C = make_float3(B.x - C.x, B.y - C.y, B.z - C.z);

		float4 cross_vec;
		cross_vec.x = A_C.y * B_C.z - B_C.y * A_C.z;
		cross_vec.y = -(A_C.x * B_C.z - B_C.x * A_C.z);
		cross_vec.z = A_C.x * B_C.y - B_C.x * A_C.y;
		if ((cross_vec.x == 0 && cross_vec.y == 0 && cross_vec.z == 0) || isnan(cross_vec.x) || isnan(cross_vec.y) || isnan(cross_vec.z)) {
			continue;
		}
		NormalizeVec3(&cross_vec);
		cross_vec.w = -(cross_vec.x * A.x + cross_vec.y * A.y + cross_vec.z * A.z);
		float temp_cost = 0.0f;
		for (int strong_index = 0; strong_index < strong_count; ++strong_index) {
			if (strong_index == a_index || strong_index == b_index || strong_index == c_index) {
				continue;
			}
			const float3& temp_point = strong_points_3d[strong_index];
			float distance = fabs(cross_vec.x * temp_point.x + cross_vec.y * temp_point.y + cross_vec.z * temp_point.z + cross_vec.w);
			temp_cost += distance;
		}
		if (temp_cost < min_cost) {
			min_cost = temp_cost;
			best_plane = cross_vec;
			has_best_plane = true;
		}
		if (min_cost == 0) {
			break;
		}
	}
	if (has_best_plane) {
		float depth = ComputeDepthfromPlaneHypothesis(camera, plane_hypotheses[center], point);
		float4 view_direction = GetViewDirection(camera, point, depth);
		float dot_product = best_plane.x * view_direction.x + best_plane.y * view_direction.y + best_plane.z * view_direction.z;
		if (dot_product > 0) {
			best_plane.x = -best_plane.x;
			best_plane.y = -best_plane.y;
			best_plane.z = -best_plane.z;
			best_plane.w = -best_plane.w;
		}

		fit_plane_hypothese[center] = best_plane;
	}
	else {
		fit_plane_hypothese[center] = make_float4(0, 0, 0, 0);
	}
}

void APD::ExportFitNormal() {
    std::shared_ptr<float4[]> fit_plane_hypotheses_host(new float4[width * height]);
    hipMemcpy(fit_plane_hypotheses_host.get(), fit_plane_hypotheses_cuda, sizeof(float4) * width * height, hipMemcpyDeviceToHost);
    cv::Mat normal_fit = cv::Mat::zeros(height, width, CV_32FC3);
    for (int r = 0; r < height; ++r) {
        for (int c = 0; c < width; ++c) {
            float4 plane = fit_plane_hypotheses_host[r * width + c];
            normal_fit.at<cv::Vec3f>(r, c) = cv::Vec3f(plane.x, plane.y, plane.z);
        }
    }
    path normal_img_path = problem.result_folder / path("fit_normal_" + std::to_string(problem.iteration) + ".png");
    ShowNormalMap(normal_img_path, normal_fit);
}

void APD::ExportAnchors() {
    path anchors_map_path = problem.result_folder / path("anchors_map.bin");
    path anchors_path = problem.result_folder / path("anchors.bin");
    WriteBinMat(anchors_map_path, anchors_map_host);
    std::shared_ptr<short2 []> anchors_host(new short2[weak_count * ANCHOR_NUM]);
    hipMemcpy(anchors_host.get(), anchors_cuda, sizeof(short2) * weak_count * ANCHOR_NUM, hipMemcpyDeviceToHost);
    ofstream out(anchors_path, std::ios_base::binary);
    int anchor_sample_num = ANCHOR_NUM;
    out.write((char*)&weak_count, sizeof(int));
    out.write((char*)&anchor_sample_num, sizeof(int));
    out.write((char*)anchors_host.get(), sizeof(short2) * weak_count * ANCHOR_NUM);
    out.close();
}

void APD::ExportNearestStrong() {
    std::shared_ptr<short2[]> nearest_strong_host(new short2[width * height]);
    hipMemcpy(nearest_strong_host.get(), weak_nearest_strong_cuda, sizeof(short2) * width * height, hipMemcpyDeviceToHost);
    cv::Mat nearest_strong_map = cv::Mat::zeros(height, width, CV_8UC3);
    std::unordered_map<int, uchar3> color_map;
    for (int r = 0; r < height; ++r) {
        for (int c = 0; c < width; ++c) {
            short2 nearest_strong = nearest_strong_host[r * width + c];
            int center = nearest_strong.y * width + nearest_strong.x;
            if (color_map.find(center) != color_map.end()) {
                nearest_strong_map.at<cv::Vec3b>(r, c) = cv::Vec3b(color_map[center].x, color_map[center].y, color_map[center].z);
            }
            else {
                uchar3 color = make_uchar3(rand() % 255, rand() % 255, rand() % 255);
                color_map[center] = color;
                nearest_strong_map.at<cv::Vec3b>(r, c) = cv::Vec3b(color.x, color.y, color.z);
            }
        }
    }
    path nearest_strong_img_path = problem.result_folder / path("nearest_strong_" + std::to_string(problem.iteration) + ".png");
    cv::imwrite(nearest_strong_img_path.string(), nearest_strong_map);
}


void APD::RunPatchMatch() {
	int BLOCK_W = 32;
	int BLOCK_H = (BLOCK_W / 2);

	dim3 grid_size_full;
	grid_size_full.x = (width + 16 - 1) / 16;
	grid_size_full.y = (height + 16 - 1) / 16;
	grid_size_full.z = 1;
	dim3 block_size_full;
	block_size_full.x = 16;
	block_size_full.y = 16;
	block_size_full.z = 1;

	dim3 grid_size_half;
	grid_size_half.x = (width + BLOCK_W - 1) / BLOCK_W;
	grid_size_half.y = ((height / 2) + BLOCK_H - 1) / BLOCK_H;
	grid_size_half.z = 1;
	dim3 block_size_half;
	block_size_half.x = BLOCK_W;
	block_size_half.y = BLOCK_H;
	block_size_half.z = 1;

	InitRandomStates << <grid_size_full, block_size_full >> > (helper_cuda);
	if (problem.params.use_APD) {
		FindNearestStrongPoint << <grid_size_full, block_size_full >> > (helper_cuda);
        GenAnchors << <grid_size_full, block_size_full >> > (helper_cuda);
		NeigbourUpdate << <grid_size_full, block_size_full >> > (helper_cuda);
	}

    if (problem.export_anchor) {
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        ExportAnchors();
    }

	RandomInitialization << <grid_size_full, block_size_full >> > (helper_cuda);

	for (int i = 0; i < params_host.max_iterations; ++i) {
		BlackPixelUpdateStrong << <grid_size_half, block_size_half >> > (i, helper_cuda);
		RedPixelUpdateStrong << <grid_size_half, block_size_half >> > (i, helper_cuda);

		if (problem.params.use_APD) {
			RANSACToGetFitPlane << <grid_size_full, block_size_full >> > (helper_cuda);
			BlackPixelUpdateWeak << <grid_size_half, block_size_half >> > (i, helper_cuda);
			RedPixelUpdateWeak << <grid_size_half, block_size_half >> > (i, helper_cuda);
		}
	}

	GetDepthandNormal << <grid_size_full, block_size_full >> > (helper_cuda);
	BlackPixelFilterStrong << <grid_size_half, block_size_half >> > (helper_cuda);
	RedPixelFilterStrong << <grid_size_half, block_size_half >> > (helper_cuda);
    DepthToWeak << <grid_size_full, block_size_full >> > (helper_cuda);
    if (problem.params.geom_consistency || problem.params.use_APD) {
        ConfidenceCompute << < grid_size_full, block_size_full >> > (helper_cuda);
    }
    LocalRefine << <grid_size_full, block_size_full >> > (helper_cuda);

	CUDA_SAFE_CALL(hipDeviceSynchronize());
	hipMemcpy(plane_hypotheses_host.get(), plane_hypotheses_cuda, sizeof(float4) * width * height, hipMemcpyDeviceToHost);
	hipMemcpy(weak_info_host.ptr<uchar>(0), weak_info_cuda, width * height * sizeof(uchar), hipMemcpyDeviceToHost);
    if (problem.params.geom_consistency || problem.params.use_APD) {
        hipMemcpy(confidence_host.ptr<uchar>(0), confidence_cuda, width * height * sizeof(uchar), hipMemcpyDeviceToHost);
    }
}